﻿#include "STPDeviceErrorHandler.cuh"

//System
#include <iostream>

__host__ inline void SuperTerrainPlus::STPCompute::cudaAssert(hipError_t cuda_code, const char* file, int line) {
	if (cuda_code != hipSuccess) {
		std::cerr << "CUDA assert: " << hipGetErrorString(cuda_code) << " in " << file << " at line " << line << std::endl;
	}
}