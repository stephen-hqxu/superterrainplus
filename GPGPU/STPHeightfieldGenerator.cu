#include "hip/hip_runtime.h"
#pragma once
#include "STPHeightfieldGenerator.cuh"

#include <memory>

__constant__ unsigned char HeightfieldSettings[sizeof(SuperTerrainPlus::STPSettings::STPHeightfieldSettings)];

using namespace SuperTerrainPlus::STPCompute;

/**
 * @brief Find the unit vector of the input vector
 * @param vec3 - Vector input
 * @return Unit vector of the input
*/
__device__ __inline__ float3 normalize3DKERNEL(float3);

/**
 * @brief Performing inverse linear interpolation for each value on the heightmap to scale it within [0,1] using CUDA kernel
 * @param minVal The mininmum value that can apperar in this height map
 * @param maxVal The maximum value that can apperar in this height map
 * @param value The input value
 * @return The interpolated value
*/
__device__ __inline__ float InvlerpKERNEL(float, float, float);

/**
 * @brief Clamp the input value with the range
 * @param val The clamping value
 * @param lower The lowest possible value
 * @param upper The highest possible value
 * @return val if [lower, upper], lower if val < lower, upper if val > upper
*/
__device__ __forceinline__ int clamp(int, int, int);

/**
 * @brief Init the hiprand generator for each thread
 * @param rng The random number generator array, it must have the same number of element as thread. e.g.,
 * generating x random number each in 1024 thread needs 1024 rng, each thread will use the same sequence.
 * @param seed The seed for each generator
*/
__global__ void curandInitKERNEL(STPHeightfieldGenerator::curandRNG*, unsigned long long);

/**
 * @brief Generate our epic height map using simplex noise function within the CUDA kernel
 * @param noise_fun - The heightfield generator that's going to use
 * @param height_storage - The pointer to a location where the heightmap will be stored
 * @param dimension - The width and height of the generated heightmap
 * @param half_dimension - Precomputed dimension/2 so the kernel don't need to repeatly compute that
 * @param offset - Controlling the offset on x, y and height offset on z
*/
__global__ void generateHeightmapKERNEL(STPSimplexNoise* const, float*, uint2, float2, float3);

/**
 * @brief Performing hydraulic erosion for the given heightmap terrain using CUDA parallel computing
 * @param height_storage Heightmap that is going to erode with raindrop
 * @param dimension The size of all maps, they must be the same
 * @param rng The random number generator map sequence, independent for each rain drop
*/
__global__ void performErosionKERNEL(float*, uint2, STPHeightfieldGenerator::curandRNG*);

/**
 * @brief Generate the normal map for the height map within kernel
 * @param heightmap - contains the height map that will be wused to generate the normal
 * @param normal_storage - normal map, will be used to store the output of the normal map
 * @param dimension - The width and height of both map
 * @return True if the normal map is successully generated without errors
*/
__global__ void generateNormalmapKERNEL(float* const, float*, uint2);

/**
 * @brief Convert _32F format to _16
 * @param input The input image, each color channel occupies 32 bit (float)
 * @param output The output image, each color channel occupies 16 bit (unsigne short int).
 * @param channel How many channel in the texture, the input and output channel will have the same number of channel
 * @return True if conversion was successful without errors
*/
__global__ void floatToshortKERNEL(const float* const, unsigned short*, uint2, unsigned int);

__host__ float* STPHeightfieldGenerator::STPHeightfieldAllocator::allocate(size_t count) {
	float* mem = nullptr;
	hipMalloc(&mem, sizeof(float) * count);
	return mem;
}

__host__ void STPHeightfieldGenerator::STPHeightfieldAllocator::deallocate(size_t count, float* ptr) {
	hipFree(ptr);
}

__host__ unsigned short* STPHeightfieldGenerator::STPImageConverterAllocator::allocate(size_t count) {
	unsigned short* mem = nullptr;
	hipMalloc(&mem, sizeof(unsigned short) * count);
	return mem;
}

__host__ void STPHeightfieldGenerator::STPImageConverterAllocator::deallocate(size_t count, unsigned short* ptr) {
	hipFree(ptr);
}

__host__ STPHeightfieldGenerator::STPHeightfieldGenerator(STPSettings::STPSimplexNoiseSettings* const noise_settings) : simplex_h(noise_settings), Noise_Settings(*noise_settings) {
	//allocating space
	hipMalloc(&this->simplex, sizeof(STPSimplexNoise));
	//copy data
	hipMemcpy(this->simplex, &simplex_h, sizeof(STPSimplexNoise), hipMemcpyHostToDevice);

	//kernel parameters
	this->numThreadperBlock_Map = dim3(32, 32);
	this->numBlock_Map = dim3(noise_settings->Dimension.x / numThreadperBlock_Map.x, noise_settings->Dimension.y / numThreadperBlock_Map.y);
	this->numThreadperBlock_Erosion = 1024;
	this->numBlock_Erosion = 0;//This will be set after user call the setErosionIterationCUDA() method
}

__host__ STPHeightfieldGenerator::~STPHeightfieldGenerator() {
	hipFree(this->simplex);
	//check if the rng has been init
	if (this->RNG_Map != nullptr) {
		hipFree(this->RNG_Map);
	}
	if (this->BiomeDictionary != nullptr) {
		hipFree(this->BiomeDictionary);
	}
}

__host__ bool STPHeightfieldGenerator::useSettings(const STPSettings::STPHeightfieldSettings* const settings) {
	//keep a local copy of the setting so device can have access to the pointer inside the class
	static std::unique_ptr<const STPSettings::STPHeightfieldSettings> stored_settings;

	if (settings == nullptr) {
		//clear memory
		stored_settings.reset();
	}
	//if memory address isn't the same
	if (stored_settings.get() != settings) {
		//validate memory
		if (!settings->validate()) {
			return false;
		}
		//replace current settings
		//deep copy the thing
		stored_settings = std::unique_ptr<const STPSettings::STPHeightfieldSettings>(new STPSettings::STPHeightfieldSettings(*settings));
	}

	return hipSuccess == hipMemcpyToSymbol(HIP_SYMBOL(HeightfieldSettings), stored_settings.get(), sizeof(STPSettings::STPHeightfieldSettings), 0ull, hipMemcpyHostToDevice);
}

__host__ bool STPHeightfieldGenerator::generateHeightfieldCUDA(STPMapStorage& args, STPGeneratorOperation operation) const {
	//check the availiability of the engine
	if (this->RNG_Map == nullptr) {
		return false;
	}
	//check the availability of biome dictionary
	/*if (this->BiomeDictionary == nullptr) {
		return false;
	}*/
	if (operation == 0u) {
		//no operation is specified, nothing can be done
		return false;
	}
	static auto isFlagged = [](STPGeneratorOperation op, STPGeneratorOperation flag) -> bool {
		return (op & flag) != 0u;
	};

	bool no_error = true;//check for error, true if all successful
	//allocating spaces for texture, storing on device
	//this is the size for a texture in one channel
	const int num_pixel = this->Noise_Settings.Dimension.x * this->Noise_Settings.Dimension.y;
	const int map_size = num_pixel * sizeof(float);
	const int map16ui_size = num_pixel * sizeof(unsigned short);
	//heightmap and normalmap
	float* heightfield_d[2] = {nullptr};
	unsigned short* heightfield_formatted_d[2] = {nullptr};
	//Retrieve all flags
	const bool flags[4] = {
		isFlagged(operation, STPHeightfieldGenerator::HeightmapGeneration),
		isFlagged(operation, STPHeightfieldGenerator::Erosion),
		isFlagged(operation, STPHeightfieldGenerator::NormalmapGeneration),
		isFlagged(operation, STPHeightfieldGenerator::Format)
	};
	//The format flags
	const bool format_flags[2] = {
		isFlagged(args.FormatHint, STPHeightfieldGenerator::FormatHeightmap),
		isFlagged(args.FormatHint, STPHeightfieldGenerator::FormatNormalmap)
	};

	//memory allocation
	//FP32
	{
		//regardlessly, we need device memory for heightmap
		std::unique_lock<std::mutex> lock(this->MapCache32F_lock);
		heightfield_d[0] = this->MapCache32F_device.allocate(map_size);
		if (flags[2] || (flags[3] && format_flags[1])) {
			//if normal map formation is enabled, we need the device memory for input as well
			heightfield_d[1] = this->MapCache32F_device.allocate(map_size * 4);
		}
	}
	//INT16
	{
		std::unique_lock<std::mutex> lock(this->MapCache16UI_lock);
		if (flags[3]) {
			if (format_flags[0]) {
				heightfield_formatted_d[0] = this->MapCache16UI_device.allocate(map16ui_size);
			}
			if (format_flags[1]) {
				heightfield_formatted_d[1] = this->MapCache16UI_device.allocate(map16ui_size * 4);
			}
		}
	}

	//setup phase
	//creating stream so cpu thread can calculate all chunks altogether
	hipStream_t stream;
	//we want the stream to not be blocked by default stream
	no_error &= hipSuccess == hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
	//Flag: HeightmapGeneration
	if (flags[0]) {
		//generate a new heightmap and store it to the output later
		generateHeightmapKERNEL << <this->numBlock_Map, this->numThreadperBlock_Map, 0, stream >> > (this->simplex, heightfield_d[0],
			this->Noise_Settings.Dimension, make_float2(1.0f * this->Noise_Settings.Dimension.x / 2.0f, 1.0f * this->Noise_Settings.Dimension.y / 2.0f), args.HeightmapOffset);
	}
	else {
		//copy heightmap from input arguments to device for later use if not generated
		no_error &= hipSuccess == hipMemcpyAsync(heightfield_d[0], *args.Heightmap32F.begin(), map_size, hipMemcpyHostToDevice, stream);
	}

	//Flag: Erosion
	if (flags[1]) {
		//erode the heightmap, either from provided heightmap or generated previously
		performErosionKERNEL << <this->numBlock_Erosion, this->numThreadperBlock_Erosion, 0, stream >> > (heightfield_d[0],
			this->Noise_Settings.Dimension, this->RNG_Map);
	}

	//Flag: Normalmap
	if (flags[2]) {
		//generate normalmap from heightmap
		generateNormalmapKERNEL << <this->numBlock_Map, this->numThreadperBlock_Map, 0, stream >> > (heightfield_d[0], heightfield_d[1],
			this->Noise_Settings.Dimension);
	}

	//Flag: Format - move STPImageConverter to here
	if (flags[3]) {
		if (format_flags[0]) {
			//format heightmap
			//heightmap will always be available
			//format heightmap
			floatToshortKERNEL << <this->numBlock_Map, this->numThreadperBlock_Map, 0, stream >> > (heightfield_d[0], heightfield_formatted_d[0],
				this->Noise_Settings.Dimension, 1);
		}

		if (format_flags[1]) {
			//format normalmap
			if (!flags[2]) {
				//normalmap generation was not enabled? we need to copy from input
				no_error &= hipSuccess == hipMemcpyAsync(heightfield_d[1], args.Normalmap32F, map16ui_size * 4, hipMemcpyHostToDevice, stream);
			}
			//if normalmap is generated, it's already available in device memory
			floatToshortKERNEL << <this->numBlock_Map, this->numThreadperBlock_Map, 0, stream >> > (heightfield_d[1], heightfield_formatted_d[1],
				this->Noise_Settings.Dimension, 4);
		}
		
	}
	
	//Store the result accordingly
	//copy the result back to the host
	//heightmap will always be available
	no_error &= hipSuccess == hipMemcpyAsync(*args.Heightmap32F.begin(), heightfield_d[0], map_size, hipMemcpyDeviceToHost, stream);
	if (flags[2]) {
		//if we have normalmap generated, also copy normalmap back to host
		no_error &= hipSuccess == hipMemcpyAsync(args.Normalmap32F, heightfield_d[1], map_size * 4, hipMemcpyDeviceToHost, stream);
	}
	//copy the formatted result if enabled
	if (flags[3]) {
		if (format_flags[0]) {
			//copy heightmap
			no_error &= hipSuccess == hipMemcpyAsync(args.Heightmap16UI, heightfield_formatted_d[0], map16ui_size, hipMemcpyDeviceToHost, stream);
		}
		if (format_flags[1]) {
			//copy normalmap
			no_error &= hipSuccess == hipMemcpyAsync(args.Normalmap16UI, heightfield_formatted_d[1], map16ui_size * 4, hipMemcpyDeviceToHost, stream);
		}
	}
	//waiting for finish
	no_error &= hipSuccess == hipStreamSynchronize(stream);

	//Finish up the rest, clear up when the device is ready
	//nullptr means not allocated
	{
		std::unique_lock<std::mutex> lock(this->MapCache32F_lock);
		if (heightfield_d[0] != nullptr) {
			this->MapCache32F_device.deallocate(map_size, heightfield_d[0]);
		}
		if (heightfield_d[1] != nullptr) {
			this->MapCache32F_device.deallocate(map_size * 4, heightfield_d[1]);
		}
	}
	{
		std::unique_lock<std::mutex> lock(this->MapCache16UI_lock);
		if (heightfield_formatted_d[0] != nullptr) {
			this->MapCache16UI_device.deallocate(map16ui_size, heightfield_formatted_d[0]);
		}
		if (heightfield_formatted_d[1] != nullptr) {
			this->MapCache16UI_device.deallocate(map16ui_size * 4, heightfield_formatted_d[1]);
		}
	}
	no_error &= hipSuccess == hipStreamDestroy(stream);

	return no_error;
}

__host__ bool STPHeightfieldGenerator::setErosionIterationCUDA(unsigned int raindrop_count) {
	//set the launch parameter
	this->numBlock_Erosion = raindrop_count / this->numThreadperBlock_Erosion;
	bool no_error = true;

	//make sure all previous takes are finished
	no_error &= hipSuccess == hipDeviceSynchronize();
	//when the raindrop count changes, we need to reallocate and regenerate the rng
	//the the number of rng = the number of the raindrop
	//such that each raindrop has independent rng
	//allocating spaces for rng storage array
	if (this->RNG_Map != nullptr) {
		//if there is an old version existing, we need to delete the old one
		no_error &= hipSuccess == hipFree(this->RNG_Map);
	}
	no_error &= hipSuccess == hipMalloc(&this->RNG_Map, sizeof(curandRNG) * raindrop_count);
	//and send to kernel
	curandInitKERNEL<<<this->numBlock_Erosion, this->numThreadperBlock_Erosion>>>(this->RNG_Map, this->Noise_Settings.Seed);
	no_error &= hipSuccess == hipDeviceSynchronize();
	//leave the result on device, and update the raindrop count
	this->NumRaindrop = raindrop_count;

	return no_error;
}

__host__ unsigned int STPHeightfieldGenerator::getErosionIteration() const {
	return this->NumRaindrop;
}

__device__ __inline__ float3 normalize3DKERNEL(float3 vec3) {
	const float length = sqrtf(powf(vec3.x, 2) + powf(vec3.y, 2) + powf(vec3.z, 2));
	return make_float3(fdividef(vec3.x, length), fdividef(vec3.y, length), fdividef(vec3.z, length));
}

__device__ __inline__ float InvlerpKERNEL(float minVal, float maxVal, float value) {
	//lerp the noiseheight to [0,1]
	return __saturatef(fdividef(value - minVal, maxVal - minVal));
}

__device__ __forceinline__ int clamp(int val, int lower, int upper) {
	return max(lower, min(val, upper));
}

__global__ void curandInitKERNEL(STPHeightfieldGenerator::curandRNG* rng, unsigned long long seed) {
	//current working index
	const unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
	//the same seed but we are looking for different sequence
	hiprand_init(seed, static_cast<unsigned long long>(index), 0, &rng[index]);
}

__global__ void generateHeightmapKERNEL(STPSimplexNoise* const noise_fun, float* height_storage,
	uint2 dimension, float2 half_dimension, float3 offset) {
	//convert constant memory to usable class
	const SuperTerrainPlus::STPSettings::STPHeightfieldSettings* const settings = reinterpret_cast<const SuperTerrainPlus::STPSettings::STPHeightfieldSettings* const>(HeightfieldSettings);

	//the current working pixel
	unsigned int x = (blockIdx.x * blockDim.x) + threadIdx.x,
		y = (blockIdx.y * blockDim.y) + threadIdx.y;
	float amplitude = 1.0f, frequency = 1.0f, noiseheight = 0.0f;
	float min = 0.0f, max = 0.0f;//The min and max indicates the range of the multi-phased simplex function, not the range of the output texture
	//multiple phases of noise
	for (int i = 0; i < settings->Octave; i++) {
		float sampleX = ((1.0 * x - half_dimension.x) + offset.x) / settings->Scale * frequency, //subtract the half width and height can make the scaling focus at the center
			sampleY = ((1.0 * y - half_dimension.y) + offset.z) / settings->Scale * frequency;//since the y is inverted we want to filp it over
		noiseheight += noise_fun->simplex2D(sampleX, sampleY) * amplitude;

		//calculate the min and max
		min -= 1.0f * amplitude;
		max += 1.0f * amplitude;
		//scale the parameters
		amplitude *= settings->Persistence;
		frequency *= settings->Lacunarity;
	}
	
	//interpolate and clamp the value within [0,1], was [min,max]
	noiseheight = InvlerpKERNEL(min, max, noiseheight + offset.y);
	//finally, output the texture
	height_storage[x + y * dimension.x] = noiseheight;//we have only allocated R32F format;
	
	return;
}

__global__ void performErosionKERNEL(float* height_storage, uint2 dimension, STPHeightfieldGenerator::curandRNG* rng) {
	//convert constant memory to usable class
	SuperTerrainPlus::STPSettings::STPRainDropSettings* const settings = (SuperTerrainPlus::STPSettings::STPRainDropSettings* const)(reinterpret_cast<const SuperTerrainPlus::STPSettings::STPHeightfieldSettings* const>(HeightfieldSettings));

	//current working index
	const unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
	//generating random location
	//first we generate the number (0.0f, 1.0f]
	float2 initPos = make_float2(hiprand_uniform(&rng[index]), hiprand_uniform(&rng[index]));
	//convert to (erode radius, dimension - erode radius - 1]
	//range: dimension - 2 * erosion radius - 1
	initPos.x *= dimension.x - 2.0f * settings->getErosionBrushRadius() - 1.0f;
	initPos.x += settings->getErosionBrushRadius();
	initPos.y *= dimension.y - 2.0f * settings->getErosionBrushRadius() - 1.0f;
	initPos.y += settings->getErosionBrushRadius();

	//spawn in the raindrop
	STPRainDrop droplet(initPos, settings->initWaterVolume, settings->initSpeed);
	//usually each droplet only does that once, rarely go beyond twice.
	//Just adding in case...
	droplet.Erode(settings, dimension, height_storage);
}

__global__ void generateNormalmapKERNEL(float* const heightmap, float* normal_storage, uint2 dimension) {
	//convert constant memory to usable class
	SuperTerrainPlus::STPSettings::STPHeightfieldSettings* const settings = reinterpret_cast<SuperTerrainPlus::STPSettings::STPHeightfieldSettings* const>(HeightfieldSettings);

	//the current working pixel
	unsigned int x = (blockIdx.x * blockDim.x) + threadIdx.x,
		y = (blockIdx.y * blockDim.y) + threadIdx.y;

	//load the cells from heightmap, remember the height map only contains one color channel
	//using Sobel fitering
	float cell[8];
	cell[0] = heightmap[clamp((x - 1), 0, dimension.x - 1) + clamp((y - 1), 0, dimension.y - 1) * dimension.x];
	cell[1] = heightmap[x + clamp((y - 1), 0, dimension.y - 1) * dimension.x];
	cell[2] = heightmap[clamp((x + 1), 0, dimension.x - 1) + clamp((y - 1), 0, dimension.y - 1) * dimension.x];
	cell[3] = heightmap[clamp((x - 1), 0, dimension.x - 1) + y * dimension.x];
	cell[4] = heightmap[clamp((x + 1), 0, dimension.x - 1) + y * dimension.x];
	cell[5] = heightmap[clamp((x - 1), 0, dimension.x - 1) + clamp((y + 1), 0, dimension.y - 1) * dimension.x];
	cell[6] = heightmap[x + clamp((y + 1), 0, dimension.y - 1) * dimension.x];
	cell[7] = heightmap[clamp((x + 1), 0, dimension.x - 1) + clamp((y + 1), 0, dimension.y - 1) * dimension.x];
	//apply the filtering kernel matrix
	float3 normal;
	normal.z = 1.0f / settings->Strength;
	normal.x = cell[0] + 2 * cell[3] + cell[5] - (cell[2] + 2 * cell[4] + cell[7]);
	normal.y = cell[0] + 2 * cell[1] + cell[2] - (cell[5] + 2 * cell[6] + cell[7]);
	//normalize
	normal = normalize3DKERNEL(normal);
	//clamp to [0,1], was [-1,1]
	normal.x = __saturatef((normal.x + 1.0f) / 2.0f);
	normal.y = __saturatef((normal.y + 1.0f) / 2.0f);
	normal.z = __saturatef((normal.z + 1.0f) / 2.0f);
	
	//copy to the output, RGBA32F
	normal_storage[(x + y * dimension.x) * 4] = normal.x;//R
	normal_storage[(x + y * dimension.x) * 4 + 1] = normal.y;//G
	normal_storage[(x + y * dimension.x) * 4 + 2] = normal.z;//B
	normal_storage[(x + y * dimension.x) * 4 + 3] = 1.0f;//A
	
	return;
}

__global__ void floatToshortKERNEL(const float* const input, unsigned short* output, uint2 dimension, unsigned int channel) {
	//current working pixel
	const unsigned int x = (blockIdx.x * blockDim.x) + threadIdx.x,
		y = (blockIdx.y * blockDim.y) + threadIdx.y,
		index = x + y * dimension.x;

	//loop through all channels and output
	for (int i = 0; i < channel; i++) {
		output[index * channel + i] = static_cast<unsigned short>(input[index * channel + i] * 65535u);
	}
}