#include "hip/hip_runtime.h"
#include "STPPermutationsGenerator.cuh"

using namespace SuperTerrainPlus::STPCompute;

__host__ STPPermutationsGenerator::STPPermutationsGenerator(unsigned long long seed, unsigned int distribution, double offset) : GRADIENT2D_SIZE(distribution) {
	//seed the engine
	STPPermutationRNG rng;
	rng.seed(seed);

	//we allocate memory in cpu and shuffle the table, then copy back to gpu
	//I was thinking about unified memory but we don't need the memory on host after the init process
	//so using pure device memory will be faster to access than unified one
	//allocation
	int* PERMUTATIONS_HOST = new int[512];
	//copy one... copy first
	copy(begin(this->INIT_TABLE), end(this->INIT_TABLE), PERMUTATIONS_HOST);
	//shuffle first, the two copy must be the same
	shuffle(PERMUTATIONS_HOST, PERMUTATIONS_HOST + 256, rng);
	//copy this the shuffled result
	copy(PERMUTATIONS_HOST, PERMUTATIONS_HOST + 256, PERMUTATIONS_HOST + 256);
		
	//now copy the host table to the device
	hipMalloc(&this->PERMUTATIONS, sizeof(int) * 512);
	hipMemcpy(this->PERMUTATIONS, PERMUTATIONS_HOST, sizeof(int) * 512, hipMemcpyHostToDevice);

	//finishing up
	delete[] PERMUTATIONS_HOST;

	//generate the gradient table
	//we are going to distribute the gradient evenly in a circle
	const double step = 360.0 / this->GRADIENT2D_SIZE * 1.0;//in degree
	double* GRADIENT2D_HOST = new double[this->GRADIENT2D_SIZE * 2];//2D so we *2
	int counter = 0;
	for (double angle = 0.0; angle < 360.0; angle += step) {//in degree
		GRADIENT2D_HOST[counter * 2] = cos(STPPermutationsGenerator::PI * (angle + offset) / 180.0);
		GRADIENT2D_HOST[counter * 2 + 1] = sin(STPPermutationsGenerator::PI * (angle + offset) / 180.0);

		counter++;
	}

	shuffle(GRADIENT2D_HOST, GRADIENT2D_HOST + this->GRADIENT2D_SIZE * 2, rng);
	//copy the host gradient to device
	hipMalloc(&this->GRADIENT2D, sizeof(double) * this->GRADIENT2D_SIZE * 2);
	hipMemcpy(this->GRADIENT2D, GRADIENT2D_HOST, sizeof(double) * this->GRADIENT2D_SIZE * 2, hipMemcpyHostToDevice);
	//finishing up
	delete[] GRADIENT2D_HOST;
}

__host__ STPPermutationsGenerator::~STPPermutationsGenerator() {
	hipFree(this->PERMUTATIONS);
	hipFree(this->GRADIENT2D);
}

__device__ int STPPermutationsGenerator::perm(int index) {
	return this->PERMUTATIONS[index];//device memory can be accessed in device directly
}

__device__ double STPPermutationsGenerator::grad2D(int index, int component) {
	return this->GRADIENT2D[index * 2 + component];
}