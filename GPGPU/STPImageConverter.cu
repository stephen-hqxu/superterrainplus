#include "hip/hip_runtime.h"
#pragma once
#include "STPImageConverter.cuh"

using namespace SuperTerrainPlus::STPCompute;


__host__ STPImageConverter::STPImageConverter(uint2 mapSize) {
	//kernel launch parameters
	this->dimension = mapSize;
	this->numThreadperBlock = dim3(32, 32);
	this->numBlock = dim3(this->dimension.x / numThreadperBlock.x, this->dimension.y / numThreadperBlock.y);
}

__host__ STPImageConverter::~STPImageConverter() {

}

__host__ bool STPImageConverter::floatToshortCUDA(const float* const input, unsigned short* output, int channel) {
	//the output, which should be _16 format
	const unsigned int num_channel = this->dimension.x * this->dimension.y * channel;
	bool no_error = true;
	//creating stream for async conversion
	hipStream_t stream;
	no_error &= hipSuccess == hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);

	//copy the host input to device
	float* input_d = nullptr;
	unsigned short* converted_d = nullptr;
	//allocating input and returning value
	no_error &= hipSuccess == hipMalloc(&input_d, sizeof(float) * num_channel);
	no_error &= hipSuccess == hipMalloc(&converted_d, sizeof(unsigned short) * num_channel);
	no_error &= hipSuccess == hipMemcpyAsync(input_d, input, sizeof(float) * num_channel, hipMemcpyHostToDevice, stream);
	
	//call the kernel function
	STPKernelLauncher::floatToshortKERNEL<<<this->numBlock, this->numThreadperBlock, 0, stream>>>(input_d, converted_d, this->dimension, channel);

	//copy the value back to cpu
	no_error &= hipSuccess == hipMemcpyAsync(output, converted_d, sizeof(unsigned short) * num_channel, hipMemcpyDeviceToHost, stream);

	//waiting until stream has finished
	no_error &= hipSuccess == hipStreamSynchronize(stream);

	//clear up and return
	no_error &= hipSuccess == hipFree(input_d);
	no_error &= hipSuccess == hipFree(converted_d);
	no_error &= hipSuccess == hipStreamDestroy(stream);
	return no_error;
}

__global__ void STPKernelLauncher::floatToshortKERNEL(const float* const input, unsigned short* output, uint2 dimension, int channel) {
	//current working pixel
	const unsigned int x = (blockIdx.x * blockDim.x) + threadIdx.x,
		y = (blockIdx.y * blockDim.y) + threadIdx.y,
		index = x + y * dimension.x;

	//loop through all channels and output
	for (int i = 0; i < channel; i++) {
		output[index * channel + i] = static_cast<unsigned short>(input[index * channel + i] * 65535u);
	}
}