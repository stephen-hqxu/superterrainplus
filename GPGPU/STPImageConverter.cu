#include "hip/hip_runtime.h"
#pragma once
#include "STPImageConverter.cuh"

using namespace SuperTerrainPlus::STPCompute;

/**
 * @brief Kernel launch and util functions
*/
namespace STPKernelLauncher {

	/**
	 * @brief Performing the conversion from _32F to _16
	 * @param input The input image, each color channel occupies 32 bit (float)
	 * @param output The output image, each color channel occupies 16 bit (unsigne short int)
	 * @param dimension The size of the map
	 * @param channel How many channel in the texture, the input and output channel will have the same number of channel
	*/
	__global__ void floatToshortKERNEL(const float* const, unsigned short*, uint2, int);

	/**
	 * @brief Convert _32F format to _16F
	 * @param input The input image, each color channel occupies 32 bit (float)
	 * @param output The output image, each color channel occupies 16 bit half float (unsigned short int will be used to interpret IEEE-754 half float format)
	 * @param dimension The size of the map
	 * @param channel How many channel in the texture, the input and output channel will have the same number of channel
	*/
	__global__ void floatTohalfKERNEL(const float* const, STPImageConverter::STPfloat16*, int2, int);
}

__host__ STPImageConverter::STPImageConverter(uint2 mapSize) {
	//kernel launch parameters
	this->dimension = mapSize;
	this->numThreadperBlock = dim3(32, 32);
	this->numBlock = dim3(this->dimension.x / numThreadperBlock.x, this->dimension.y / numThreadperBlock.y);
}

__host__ STPImageConverter::~STPImageConverter() {

}

__host__ bool STPImageConverter::floatToshortCUDA(const float* const input, unsigned short* output, int channel) const {
	//the output, which should be _16 format
	const unsigned int num_channel = this->dimension.x * this->dimension.y * channel;
	bool no_error = true;
	//creating stream for async conversion
	hipStream_t stream;
	no_error &= hipSuccess == hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);

	//copy the host input to device
	float* input_d = nullptr;
	unsigned short* converted_d = nullptr;
	//allocating input and returning value
	no_error &= hipSuccess == hipMalloc(&input_d, sizeof(float) * num_channel);
	no_error &= hipSuccess == hipMalloc(&converted_d, sizeof(unsigned short) * num_channel);
	no_error &= hipSuccess == hipMemcpyAsync(input_d, input, sizeof(float) * num_channel, hipMemcpyHostToDevice, stream);
	
	//call the kernel function
	STPKernelLauncher::floatToshortKERNEL<<<this->numBlock, this->numThreadperBlock, 0, stream>>>(input_d, converted_d, this->dimension, channel);

	//copy the value back to cpu
	no_error &= hipSuccess == hipMemcpyAsync(output, converted_d, sizeof(unsigned short) * num_channel, hipMemcpyDeviceToHost, stream);

	//waiting until stream has finished
	no_error &= hipSuccess == hipStreamSynchronize(stream);

	//clear up and return
	no_error &= hipSuccess == hipFree(input_d);
	no_error &= hipSuccess == hipFree(converted_d);
	no_error &= hipSuccess == hipStreamDestroy(stream);
	return no_error;
}

__global__ void STPKernelLauncher::floatToshortKERNEL(const float* const input, unsigned short* output, uint2 dimension, int channel) {
	//current working pixel
	const unsigned int x = (blockIdx.x * blockDim.x) + threadIdx.x,
		y = (blockIdx.y * blockDim.y) + threadIdx.y,
		index = x + y * dimension.x;

	//loop through all channels and output
	for (int i = 0; i < channel; i++) {
		output[index * channel + i] = static_cast<unsigned short>(input[index * channel + i] * 65535u);
	}
}