#include "hip/hip_runtime.h"
#pragma once
#include <SuperTerrain+/GPGPU/STPHeightfieldGenerator.cuh>

//Simulator
#include <SuperTerrain+/GPGPU/FreeSlip/STPFreeSlipManager.cuh>
#include <SuperTerrain+/GPGPU/STPRainDrop.cuh>

#include <SuperTerrain+/Utility/STPDeviceErrorHandler.h>
#include <SuperTerrain+/Utility/Exception/STPInvalidEnvironment.h>

#include <type_traits>
#include <memory>
//CUDA Device Parameters
#include <>

//Template definition for the smart device memory
#include <SuperTerrain+/Utility/STPSmartDeviceMemory.tpp>

using namespace SuperTerrainPlus::STPCompute;

using std::vector;
using std::mutex;
using std::unique_lock;
using std::unique_ptr;
using std::optional;
using std::move;
using std::make_unique;
using std::current_exception;
using std::rethrow_exception;

//GLM
#include <glm/vec3.hpp>
#include <glm/geometric.hpp>

using glm::ivec2;
using glm::uvec2;
using glm::vec2;
using glm::vec3;

enum class STPHeightfieldGenerator::STPEdgeArrangement : unsigned char {
	TOP_LEFT_CORNER = 0x00u,
	TOP_RIGHT_CORNER = 0x01u,
	BOTTOM_LEFT_CORNER = 0x02u,
	BOTTOM_RIGHT_CORNER = 0x03u,
	TOP_HORIZONTAL_STRIP = 0x10u,
	BOTTOM_HORIZONTAL_STRIP = 0x11u,
	LEFT_VERTICAL_STRIP = 0x12u,
	RIGHT_VERTICAL_STRIP = 0x13u,
	NOT_AN_EDGE = 0xffu
};

/**
 * @brief Init the hiprand generator for each thread
 * @param rng The random number generator array, it must have the same number of element as thread. e.g.,
 * generating x random number each in 1024 thread needs 1024 rng, each thread will use the same sequence.
 * @param seed The seed for each generator
 * @param raindrop_count The expected number of raindrop, so does the total number of RNG to init
*/
__global__ void curandInitKERNEL(STPHeightfieldGenerator::curandRNG*, unsigned long long, unsigned int);

/**
 * @brief Performing hydraulic erosion for the given heightmap terrain using CUDA parallel computing
 * @param height_storage The floating point heightmap with global-local convertion management
 * @param heightfield_settings - The settings to use to generate heightmap
 * @param rng The random number generator map sequence, independent for each rain drop
*/
__global__ void performErosionKERNEL(STPFreeSlipFloatManager, const SuperTerrainPlus::STPEnvironment::STPHeightfieldSetting*, STPHeightfieldGenerator::curandRNG*);

/**
 * @brief Generate the normal map for the height map within kernel, and combine two maps into a rendering buffer
 * @param heightmap - contains the floating point height map that will be used to generate the normalmap, with free-slip manager
 * @param strength - The strenghth of the generated normal map
 * @param heightfield - will be used to store the output of the normal map in RGB channel, heightmap will be copied to A channel
*/
__global__ void generateRenderingBufferKERNEL(STPFreeSlipFloatManager, float, unsigned short*);


__host__ STPHeightfieldGenerator::STPHeightfieldGenerator(const STPEnvironment::STPChunkSetting& chunk_settings, const STPEnvironment::STPHeightfieldSetting& heightfield_settings,
	const STPDiversityGenerator& diversity_generator, unsigned int hint_level_of_concurrency)
	: generateHeightmap(diversity_generator), Heightfield_Setting_h(heightfield_settings), 
	FreeSlipTable(chunk_settings.FreeSlipChunk, chunk_settings.MapSize) {
	if (!chunk_settings.validate()) {
		throw STPException::STPInvalidEnvironment("Values from STPChunkSetting are not validated");
	}
	if (!heightfield_settings.validate()) {
		throw STPException::STPInvalidEnvironment("Values from STPHeightfieldSetting are not validated");
	}

	const unsigned int num_pixel = this->FreeSlipTable.getDimension().x * this->FreeSlipTable.getDimension().y,
		num_freeslip_pixel = this->FreeSlipTable.getFreeSlipRange().x * this->FreeSlipTable.getFreeSlipRange().y;
	this->TextureBufferAttr.TexturePixel = num_pixel;

	//allocating space
	//heightfield settings
	this->Heightfield_Setting_d = STPSmartDeviceMemory::makeDevice<STPEnvironment::STPHeightfieldSetting>();
	STPcudaCheckErr(hipMemcpy(this->Heightfield_Setting_d.get(), &this->Heightfield_Setting_h, sizeof(STPEnvironment::STPHeightfieldSetting), hipMemcpyHostToDevice));
	
	//create memory pool
	hipMemPoolProps pool_props = { };
	pool_props.allocType = hipMemAllocationTypePinned;
	pool_props.location.id = 0;
	pool_props.location.type = hipMemLocationTypeDevice;
	pool_props.handleTypes = hipMemHandleTypeNone;
	STPcudaCheckErr(hipMemPoolCreate(&this->MapCacheDevice, &pool_props));
	//TODO: smartly determine the average memory pool size
	cuuint64_t release_thres = (sizeof(float) + sizeof(unsigned short) * 4u) * num_freeslip_pixel * hint_level_of_concurrency;
	STPcudaCheckErr(hipMemPoolSetAttribute(this->MapCacheDevice, hipMemPoolAttrReleaseThreshold, &release_thres));
	this->TextureBufferAttr.DeviceMemPool = this->MapCacheDevice;

	//init erosion
	this->setErosionIterationCUDA();
	//init edge table
	this->initEdgeArrangementTable();
}

__host__ STPHeightfieldGenerator::~STPHeightfieldGenerator() {
	STPcudaCheckErr(hipMemPoolDestroy(this->MapCacheDevice));
	//device ptrs are deleted with custom deleter
}

__host__ void STPHeightfieldGenerator::operator()(STPMapStorage& args, STPGeneratorOperation operation) const {
	//check the availiability of the engine
	if (this->RNG_Map == nullptr) {
		return;
	}
	if (operation == 0u) {
		//no operation is specified, nothing can be done
		return;
	}

	int Mingridsize, gridsize, blocksize;
	//Retrieve all flags
	auto isFlagged = []__host__(STPGeneratorOperation op, STPGeneratorOperation flag) -> bool {
		return (op & flag) != 0u;
	};
	const bool flag[3] = {
		isFlagged(operation, STPHeightfieldGenerator::HeightmapGeneration),
		isFlagged(operation, STPHeightfieldGenerator::Erosion),
		isFlagged(operation, STPHeightfieldGenerator::RenderingBufferGeneration)
	};
	STPcudaCheckErr(hipSetDevice(0));

	//creating stream so cpu thread can calculate all chunks altogether
	//if exception is thrown during exception, stream will be the last object to be deleted, automatically
	optional<STPSmartStream> stream_buffer;
	hipStream_t stream;
	//limit the scope for std::optional to control the destructor call
	{
		//heightmap
		optional<STPFreeSlipFloatTextureBuffer> heightmap_buffer;
		optional<STPFreeSlipRenderTextureBuffer> heightfield_buffer;
		optional<STPFreeSlipGenerator::STPFreeSlipFloatManagerAdaptor> heightmap_adaptor;
		//biomemap
		optional<STPFreeSlipSampleTextureBuffer> biomemap_buffer;
		optional<STPFreeSlipGenerator::STPFreeSlipSampleManagerAdaptor> biomemap_adaptor;

		//setup phase
		//we want the stream to not be blocked by default stream
		{
			unique_lock<mutex> stream_lock(this->StreamPool_lock);
			if (this->StreamPool.empty()) {
				//create a new stream
				stream_buffer.emplace(hipStreamNonBlocking);
			}
			else {
				//grab an exisiting stream
				stream_buffer.emplace(move(this->StreamPool.front()));
				this->StreamPool.pop();
			}
		}
		stream = *stream_buffer;

		//Flag: HeightmapGeneration
		if (flag[0]) {
			//generate a new heightmap using diversity generator and store it to the output later
			//copy biome map to device, and allocate heightmap
			STPFreeSlipFloatTextureBuffer::STPFreeSlipTextureData heightmap_data{ 1u, STPFreeSlipFloatTextureBuffer::STPFreeSlipTextureData::STPMemoryMode::WriteOnly, stream };
			STPFreeSlipSampleTextureBuffer::STPFreeSlipTextureData biomemap_data{ 1u, STPFreeSlipSampleTextureBuffer::STPFreeSlipTextureData::STPMemoryMode::ReadOnly, stream };

			heightmap_buffer.emplace(args.Heightmap32F, heightmap_data, this->TextureBufferAttr);
			biomemap_buffer.emplace(args.Biomemap, biomemap_data, this->TextureBufferAttr);
			biomemap_adaptor.emplace(this->FreeSlipTable(*biomemap_buffer));

			this->generateHeightmap(*heightmap_buffer, *biomemap_adaptor, args.HeightmapOffset, stream);
		}
		else {
			//no generation, use existing
			STPFreeSlipFloatTextureBuffer::STPFreeSlipTextureData heightmap_data{ 1u, STPFreeSlipFloatTextureBuffer::STPFreeSlipTextureData::STPMemoryMode::ReadWrite, stream };
			heightmap_buffer.emplace(args.Heightmap32F, heightmap_data, this->TextureBufferAttr);
		}

		if (flag[1] || flag[2]) {
			//prepare free-slip utility for heightmap
			heightmap_adaptor.emplace(this->FreeSlipTable(*heightmap_buffer));
			STPFreeSlipFloatManager heightmap_slip = (*heightmap_adaptor)(STPFreeSlipLocation::DeviceMemory);

			//Flag: Erosion
			if (flag[1]) {
				const unsigned erosionBrushCache_size = this->Heightfield_Setting_h.getErosionBrushSize() * (sizeof(int) + sizeof(float));
				STPcudaCheckErr(hipOccupancyMaxPotentialBlockSize(&Mingridsize, &blocksize, &performErosionKERNEL, erosionBrushCache_size));
				gridsize = (this->Heightfield_Setting_h.RainDropCount + blocksize - 1) / blocksize;

				//erode the heightmap, either from provided heightmap or generated previously
				performErosionKERNEL << <gridsize, blocksize, erosionBrushCache_size, stream >> > (heightmap_slip, this->Heightfield_Setting_d.get(), this->RNG_Map.get());
				STPcudaCheckErr(hipGetLastError());
			}

			//Flag: RenderingBufferGeneration
			if (flag[2]) {
				//allocate formation memory
				STPFreeSlipRenderTextureBuffer::STPFreeSlipTextureData heightfield_data{ 4u, STPFreeSlipRenderTextureBuffer::STPFreeSlipTextureData::STPMemoryMode::WriteOnly, stream };
				heightfield_buffer.emplace(args.Heightfield16UI, heightfield_data, this->TextureBufferAttr);

				auto det_cacheSize = []__host__ __device__(int blockSize) -> size_t {
					return (blockSize + 2u) * sizeof(float);
				};
				STPcudaCheckErr(hipOccupancyMaxPotentialBlockSizeVariableSMem(&Mingridsize, &blocksize, &generateRenderingBufferKERNEL, det_cacheSize));
				const uvec2 DimblockSize(32u, static_cast<unsigned int>(blocksize) / 32u),
					DimgridSize = (this->FreeSlipTable.getFreeSlipRange() + DimblockSize - 1u) / DimblockSize;

				//get free-slip util and memory
				unsigned short* heightfield_formatted_d = (*heightfield_buffer)(STPFreeSlipLocation::DeviceMemory);
				if (args.Heightfield16UI.size() > 1u) {
					//no need to do copy if freeslip is not enabled
					//this is the way to make sure normalmap is seamless, since the border is already in-sync with other chunks
					this->copyNeighbourEdgeOnly(heightfield_formatted_d, args.Heightfield16UI, this->TextureBufferAttr.TexturePixel, stream);
				}
				//generate normalmap from heightmap and format into rendering buffer
				const uvec2 cacheBlockSize = DimblockSize + 2u;
				const unsigned int cacheSize = cacheBlockSize.x * cacheBlockSize.y * sizeof(float);
				generateRenderingBufferKERNEL << <dim3(DimgridSize.x, DimgridSize.y), dim3(DimblockSize.x, DimblockSize.y), cacheSize, stream >> > (heightmap_slip, this->Heightfield_Setting_h.Strength, heightfield_formatted_d);
				STPcudaCheckErr(hipGetLastError());
			}
		}

		//Store the result accordingly
		//copy the result back to the host
		//it will call the destructor in texture buffer (optional calls it when goes out of scope), and result will be copied back using CUDA stream
		//this operation is stream ordered
	}

	//waiting for finish before release the stream back to the pool
	STPcudaCheckErr(hipStreamSynchronize(stream));
	{
		unique_lock<mutex> stream_lock(this->StreamPool_lock);
		this->StreamPool.emplace(move(*stream_buffer));
	}
}

__host__ void STPHeightfieldGenerator::setErosionIterationCUDA() {
	const unsigned int raindrop_count = this->Heightfield_Setting_h.RainDropCount;
	//determine launch parameters
	int Mingridsize, gridsize, blocksize;
	STPcudaCheckErr(hipOccupancyMaxPotentialBlockSize(&Mingridsize, &blocksize, &curandInitKERNEL));
	gridsize = (this->Heightfield_Setting_h.RainDropCount + blocksize - 1) / blocksize;

	//make sure all previous takes are finished
	STPcudaCheckErr(hipDeviceSynchronize());
	//when the raindrop count changes, we need to reallocate and regenerate the rng
	//the the number of rng = the number of the raindrop
	//such that each raindrop has independent rng
	//allocating spaces for rng storage array
	this->RNG_Map = STPSmartDeviceMemory::makeDevice<curandRNG[]>(raindrop_count);
	//and send to kernel
	curandInitKERNEL << <gridsize, blocksize >> > (this->RNG_Map.get(), this->Heightfield_Setting_h.Seed, raindrop_count);
	STPcudaCheckErr(hipGetLastError());
	STPcudaCheckErr(hipDeviceSynchronize());
}

__host__ void STPHeightfieldGenerator::copyNeighbourEdgeOnly(unsigned short* device, const vector<unsigned short*>& source, size_t element_count, hipStream_t stream) const {
	typedef STPHeightfieldGenerator::STPEdgeArrangement STPEA;
	const uvec2& dimension = this->FreeSlipTable.getDimension();
	const unsigned int one_pixel_size = 4u * sizeof(unsigned short);
	const unsigned int pitch = dimension.x * one_pixel_size;
	const unsigned int horizontal_stripe_size = dimension.x * one_pixel_size;
	//we want to cut down the number of copy of column major matrix due to concern about cache
	/**
	* Out copy pattern:				It's more efficient than:
	 ---------------------			+-------------------+
	 |                   |			|                   |
	 |                   |			|                   |
	 |                   |			|                   |
	 |                   |			|                   |
	 ---------------------			+-------------------+
	*/
	//address offset of those situations, eliminate overlap of pixels
	const unsigned int right_vertical_wholerow = (dimension.x - 1u) * 4u,
		left_vertical_skipfirstrow = right_vertical_wholerow + 4u,
		right_vertical_skipfirstrow = left_vertical_skipfirstrow + right_vertical_wholerow,
		bottom_horizontal = dimension.x * 4u * (dimension.y - 1u);

	for (int i = 0; i < source.size(); i++) {
		auto perform_copy = [device, stream, map = source[i], pitch]__host__(size_t start, size_t width_byte, size_t height) -> void {
			STPcudaCheckErr(hipMemcpy2DAsync(device + start, pitch, map + start, pitch, width_byte, height, hipMemcpyHostToDevice, stream));
		};

		switch (this->EdgeArrangementTable[i]) {
		case STPEA::TOP_LEFT_CORNER:
			//------------
			//|
			//|
			//|
			//|
			perform_copy(0u, horizontal_stripe_size, 1u);
			perform_copy(left_vertical_skipfirstrow, one_pixel_size, dimension.y - 1u);
			break;
		case STPEA::TOP_RIGHT_CORNER:
			//-------------
			//            |
			//            |
			//            |
			//            |
			perform_copy(0u, horizontal_stripe_size, 1u);
			perform_copy(right_vertical_skipfirstrow, one_pixel_size, dimension.y - 1u);
			break;
		case STPEA::BOTTOM_LEFT_CORNER:
			//|
			//|
			//|
			//|
			//-------------
			perform_copy(bottom_horizontal, horizontal_stripe_size, 1u);
			perform_copy(0u, one_pixel_size, dimension.y - 1u);
			break;
		case STPEA::BOTTOM_RIGHT_CORNER:
			//             |
			//             |
			//             |
			//             |
			//--------------
			perform_copy(bottom_horizontal, horizontal_stripe_size, 1u);
			perform_copy(right_vertical_wholerow, one_pixel_size, dimension.y - 1u);
			break;
		case STPEA::TOP_HORIZONTAL_STRIP:
			//--------------
			//
			//
			//
			//
			perform_copy(0u, horizontal_stripe_size, 1u);
			break;
		case STPEA::BOTTOM_HORIZONTAL_STRIP:
			//
			//
			//
			//
			//--------------
			perform_copy(bottom_horizontal, horizontal_stripe_size, 1u);
			break;
		case STPEA::LEFT_VERTICAL_STRIP:
			//|
			//|
			//|
			//|
			//|
			perform_copy(0u, one_pixel_size, dimension.y);
			break;
		case STPEA::RIGHT_VERTICAL_STRIP:
			//             |
			//             |
			//             |
			//             |
			//             |
			perform_copy(right_vertical_wholerow, one_pixel_size, dimension.y);
			break;
		default:
			//skip every non-edge chunk
			break;
		}
		device += element_count * 4u;
	}
}

__host__ void STPHeightfieldGenerator::initEdgeArrangementTable() {
	typedef STPHeightfieldGenerator::STPEdgeArrangement STPEA;
	const uvec2& freeslip_chunk = this->FreeSlipTable.getFreeSlipChunk();
	const unsigned int num_chunk = freeslip_chunk.x * freeslip_chunk.y;
	if (num_chunk == 1u) {
		//if freeslip logic is not turned on, there's no need to do copy
		//since edge is calculated by neighbour chunks, but without freeslip there's no "other" chunks
		//so the chunk itself needs to compute the border during rendering buffer generation
		return;
	}

	//allocate space
	this->EdgeArrangementTable = make_unique<STPEA[]>(num_chunk);
	for (unsigned int chunkID = 0u; chunkID < num_chunk; chunkID++) {
		STPEA& current_entry = this->EdgeArrangementTable[chunkID];
		const uvec2 chunkCoord(chunkID % freeslip_chunk.x, chunkID / freeslip_chunk.x);

		//some basic boolean logic to determine our "frame"
		if (chunkCoord.x == 0u) {
			if (chunkCoord.y == 0u) {
				current_entry = STPEA::TOP_LEFT_CORNER;
				continue;
			}
			if (chunkCoord.y == freeslip_chunk.y - 1u) {
				current_entry = STPEA::BOTTOM_LEFT_CORNER;
				continue;
			}
			current_entry = STPEA::LEFT_VERTICAL_STRIP;
			continue;
		}
		if (chunkCoord.x == freeslip_chunk.x - 1u) {
			if (chunkCoord.y == 0u) {
				current_entry = STPEA::TOP_RIGHT_CORNER;
				continue;
			}
			if (chunkCoord.y == freeslip_chunk.y - 1u) {
				current_entry = STPEA::BOTTOM_RIGHT_CORNER;
				continue;
			}
			current_entry = STPEA::RIGHT_VERTICAL_STRIP;
			continue;
		}

		if (chunkCoord.y == 0u) {
			current_entry = STPEA::TOP_HORIZONTAL_STRIP;
			continue;
		}
		if (chunkCoord.y == freeslip_chunk.y - 1u) {
			current_entry = STPEA::BOTTOM_HORIZONTAL_STRIP;
			continue;
		}

		//we can safely ignore edge that's not an edge chunk
		current_entry = STPEA::NOT_AN_EDGE;
	}
}

__global__ void curandInitKERNEL(STPHeightfieldGenerator::curandRNG* rng, unsigned long long seed, unsigned int raindrop_count) {
	//current working index
	const unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index >= raindrop_count) {
		return;
	}

	//the same seed but we are looking for different sequence
	hiprand_init(seed, static_cast<unsigned long long>(index), 0, &rng[index]);
}

__global__ void performErosionKERNEL(STPFreeSlipFloatManager heightmap_storage, const SuperTerrainPlus::STPEnvironment::STPHeightfieldSetting* heightfield_settings, STPHeightfieldGenerator::curandRNG* rng) {
	//current working index
	const unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index >= heightfield_settings->RainDropCount) {
		return;
	}

	//convert to (base, dimension - 1]
	//range: dimension
	//Generate the raindrop at the central chunk only
	__shared__ uvec2 base;
	__shared__ uvec2 range;
	if (threadIdx.x == 0u) {
		const uvec2& dimension = heightmap_storage.Data->Dimension;

		base = dimension - 1u,
		range = (heightmap_storage.Data->FreeSlipChunk / 2u) * dimension;
	}
	__syncthreads();

	//generating random location
	//first we generate the number (0.0f, 1.0f]
	vec2 initPos = vec2(hiprand_uniform(&rng[index]), hiprand_uniform(&rng[index]));
	//range convertion
	initPos *= base;
	initPos += range;

	//spawn in the raindrop
	STPRainDrop droplet(initPos, heightfield_settings->initWaterVolume, heightfield_settings->initSpeed);
	droplet.Erode(static_cast<const SuperTerrainPlus::STPEnvironment::STPRainDropSetting*>(heightfield_settings), heightmap_storage);
}

__global__ void generateRenderingBufferKERNEL(STPFreeSlipFloatManager heightmap, float strength, unsigned short* heightfield) {
	//the current working pixel
	const uvec2 block = uvec2(blockIdx.x * blockDim.x, blockIdx.y * blockDim.y),
		local_thread = uvec2(threadIdx.x, threadIdx.y),
		thread = local_thread + block;
	const unsigned int threadperblock = blockDim.x * blockDim.y;
	const uvec2& freeslip_range = heightmap.Data->FreeSlipRange;
	if (thread.x >= freeslip_range.x || thread.y >= freeslip_range.y) {
		return;
	}

	const uvec2& dimension = heightmap.Data->FreeSlipRange;
	auto float2short = []__device__(float input) -> unsigned short {
		return static_cast<unsigned short>(input * 0xFFFFu);
	};

	//Cache heightmap the current thread block needs since each pixel is accessed upto 9 times.
	extern __shared__ float heightmapCache[];
	//each thread needs to access a 3x3 matrix around the current pixel, so we need to take the edge into consideration
	const uvec2 cacheSize = uvec2(blockDim.x, blockDim.y) + 2u;
	unsigned int iteration = 0u;
	const unsigned int cacheSize_total = cacheSize.x * cacheSize.y;

	while (iteration < cacheSize_total) {
		const unsigned int cacheIdx = (threadIdx.x + blockDim.x * threadIdx.y) + iteration;
		const uvec2 worker = block + uvec2(cacheIdx % cacheSize.x, cacheIdx / cacheSize.x);
		//worker index may be zero, and 0u - 1u will become UINT32_MAX, so we should cast it to int and it will be come -1 thus correctly clampped
		const uvec2 clamppeWorkerIdx = static_cast<uvec2>(glm::clamp(static_cast<ivec2>(worker) - 1, ivec2(0), static_cast<ivec2>(dimension - 1u)));
		const unsigned int workerIdx = clamppeWorkerIdx.x + clamppeWorkerIdx.y * dimension.x;

		if (cacheIdx < cacheSize_total) {
			//make sure index don't get out of bound
			//start caching from (x-1, y-1) until (x+1, y+1)
			heightmapCache[cacheIdx] = heightmap[workerIdx];
		}
		//warp around to reuse some threads to finish all compute
		iteration += threadperblock;
	}
	__syncthreads();

	if ((heightmap.Data->FreeSlipChunk.x * heightmap.Data->FreeSlipChunk.y) > 1 && 
		(thread.x == 0 || thread.y == 0 || thread.x == freeslip_range.x - 1 || thread.y == freeslip_range.y - 1)) {
		//if freeslip is not turned on, we need to calculate the edge pixel for this chunk
		//otherwise, do not touch the border pixel since border pixel is calculated seamlessly by other chunks
		return;
	}

	//load the cells from heightmap, remember the height map only contains one color channel
	//using Sobel fitering
	//Cache index
	const uvec2 cache = local_thread + 1u;
	//no need to pass by reference since vec2 type a big as a pointer
	auto loadCache = [cacheIdx = cache, horizontalCacheSize = cacheSize.x]__device__(float* cache, ivec2 offset) -> float {
		const uvec2 coord = static_cast<uvec2>(static_cast<ivec2>(cacheIdx) + offset);
		return cache[coord.x + coord.y * horizontalCacheSize];
	};

	float cell[8];
	cell[0] = loadCache(heightmapCache, ivec2(-1, -1));
	cell[1] = loadCache(heightmapCache, ivec2(0, -1));
	cell[2] = loadCache(heightmapCache, ivec2(+1, -1));
	cell[3] = loadCache(heightmapCache, ivec2(-1, 0));
	cell[4] = loadCache(heightmapCache, ivec2(+1, 0));
	cell[5] = loadCache(heightmapCache, ivec2(-1, +1));
	cell[6] = loadCache(heightmapCache, ivec2(0, +1));
	cell[7] = loadCache(heightmapCache, ivec2(+1, +1));
	//apply the filtering kernel matrix
	vec3 normal;
	normal.z = 1.0f / strength;
	normal.x = cell[0] + 2 * cell[3] + cell[5] - (cell[2] + 2 * cell[4] + cell[7]);
	normal.y = cell[0] + 2 * cell[1] + cell[2] - (cell[5] + 2 * cell[6] + cell[7]);
	//normalize
	normal = glm::normalize(normal);
	//clamp to [0,1], was [-1,1]
	normal = (glm::clamp(normal, -1.0f, 1.0f) + 1.0f) / 2.0f;
	
	//copy to the output, RGBA32F
	const unsigned int index = heightmap(thread.x + thread.y * dimension.x);
	heightfield[index * 4] = float2short(normal.x);//R
	heightfield[index * 4 + 1] = float2short(normal.y);//G
	heightfield[index * 4 + 2] = float2short(normal.z);//B
	heightfield[index * 4 + 3] = float2short(heightmapCache[cache.x + cache.y * cacheSize.x]);//A
}