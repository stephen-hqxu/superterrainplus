#include "hip/hip_runtime.h"
#pragma once
#include <GPGPU/STPHeightfieldGenerator.cuh>

//Simulator
#include <GPGPU/STPFreeSlipManager.cuh>
#include <GPGPU/STPRainDrop.cuh>

#define STP_EXCEPTION_ON_ERROR
#include <Utility/STPDeviceErrorHandler.h>

#include <type_traits>
#include <memory>
//CUDA Device Parameters
#include <>

using namespace SuperTerrainPlus::STPCompute;

using std::vector;
using std::mutex;
using std::unique_lock;
using std::unique_ptr;
using std::optional;
using std::move;
using std::make_unique;

//GLM
#include <glm/vec3.hpp>
#include <glm/geometric.hpp>

using glm::uvec2;
using glm::vec2;
using glm::vec3;

enum class STPHeightfieldGenerator::STPEdgeArrangement : unsigned char {
	TOP_LEFT_CORNER = 0x00u,
	TOP_RIGHT_CORNER = 0x01u,
	BOTTOM_LEFT_CORNER = 0x02u,
	BOTTOM_RIGHT_CORNER = 0x03u,
	TOP_HORIZONTAL_STRIP = 0x10u,
	BOTTOM_HORIZONTAL_STRIP = 0x11u,
	LEFT_VERTICAL_STRIP = 0x12u,
	RIGHT_VERTICAL_STRIP = 0x13u,
	NOT_AN_EDGE = 0xffu
};

/**
 * @brief Init the hiprand generator for each thread
 * @param rng The random number generator array, it must have the same number of element as thread. e.g.,
 * generating x random number each in 1024 thread needs 1024 rng, each thread will use the same sequence.
 * @param seed The seed for each generator
 * @param raindrop_count The expected number of raindrop, so does the total number of RNG to init
*/
__global__ void curandInitKERNEL(STPHeightfieldGenerator::curandRNG*, unsigned long long, unsigned int);

/**
 * @brief Performing hydraulic erosion for the given heightmap terrain using CUDA parallel computing
 * @param height_storage The floating point heightmap with global-local convertion management
 * @param heightfield_settings - The settings to use to generate heightmap
 * @param rng The random number generator map sequence, independent for each rain drop
*/
__global__ void performErosionKERNEL(STPFreeSlipFloatManager, const SuperTerrainPlus::STPEnvironment::STPHeightfieldSetting*, STPHeightfieldGenerator::curandRNG*);

/**
 * @brief Generate the normal map for the height map within kernel, and combine two maps into a rendering buffer
 * @param heightmap - contains the floating point height map that will be used to generate the normalmap, with free-slip manager
 * @param strength - The strenghth of the generated normal map
 * @param heightfield - will be used to store the output of the normal map in RGB channel, heightmap will be copied to A channel
*/
__global__ void generateRenderingBufferKERNEL(STPFreeSlipFloatManager, float, unsigned short*);

template<typename T>
void STPHeightfieldGenerator::STPDeviceDeleter<T>::operator()(T* ptr) const {
	STPcudaCheckErr(hipFree(ptr));
}

__host__ STPHeightfieldGenerator::STPHeightfieldGenerator(const STPEnvironment::STPChunkSetting& chunk_settings, const STPEnvironment::STPHeightfieldSetting& heightfield_settings,
	const STPDiversityGenerator& diversity_generator, unsigned int hint_level_of_concurrency)
	: generateHeightmap(diversity_generator), Heightfield_Setting_h(heightfield_settings), 
	FreeSlipTable(chunk_settings.FreeSlipChunk, chunk_settings.MapSize) {
	const unsigned int num_pixel = this->FreeSlipTable.getDimension().x * this->FreeSlipTable.getDimension().y,
		num_freeslip_pixel = this->FreeSlipTable.getFreeSlipRange().x * this->FreeSlipTable.getFreeSlipRange().y;
	this->TextureBufferAttr.TexturePixel = num_pixel;

	//allocating space
	//heightfield settings
	STPEnvironment::STPHeightfieldSetting* hfs_cache;
	STPcudaCheckErr(hipMalloc(&hfs_cache, sizeof(STPEnvironment::STPHeightfieldSetting)));
	STPcudaCheckErr(hipMemcpy(hfs_cache, &this->Heightfield_Setting_h, sizeof(STPEnvironment::STPHeightfieldSetting), hipMemcpyHostToDevice));
	this->Heightfield_Setting_d = unique_ptr_d<STPEnvironment::STPHeightfieldSetting>(hfs_cache);
	
	//create memory pool
	hipMemPoolProps pool_props = { };
	pool_props.allocType = hipMemAllocationTypePinned;
	pool_props.location.id = 0;
	pool_props.location.type = hipMemLocationTypeDevice;
	pool_props.handleTypes = hipMemHandleTypeNone;
	STPcudaCheckErr(hipMemPoolCreate(&this->MapCacheDevice, &pool_props));
	//TODO: smartly determine the average memory pool size
	cuuint64_t release_thres = (sizeof(float) + sizeof(unsigned short) * 4u) * num_freeslip_pixel * hint_level_of_concurrency;
	STPcudaCheckErr(hipMemPoolSetAttribute(this->MapCacheDevice, hipMemPoolAttrReleaseThreshold, &release_thres));
	this->TextureBufferAttr.DeviceMemPool = this->MapCacheDevice;

	//init erosion
	this->setErosionIterationCUDA();
	//init edge table
	this->initEdgeArrangementTable();
}

__host__ STPHeightfieldGenerator::~STPHeightfieldGenerator() {
	STPcudaCheckErr(hipMemPoolDestroy(this->MapCacheDevice));
	//device ptrs are deleted with custom deleter
}

__host__ void STPHeightfieldGenerator::operator()(STPMapStorage& args, STPGeneratorOperation operation) const {
	//check the availiability of the engine
	if (this->RNG_Map == nullptr) {
		return;
	}
	if (operation == 0u) {
		//no operation is specified, nothing can be done
		return;
	}

	std::exception_ptr exp;
	int Mingridsize, gridsize, blocksize;
	//heightmap
	optional<STPFreeSlipFloatTextureBuffer> heightmap_buffer;
	optional<STPFreeSlipRenderTextureBuffer> heightfield_buffer;
	optional<STPFreeSlipGenerator::STPFreeSlipFloatManagerAdaptor> heightmap_adaptor;
	//biomemap
	optional<STPFreeSlipSampleTextureBuffer> biomemap_buffer;
	optional<STPFreeSlipGenerator::STPFreeSlipSampleManagerAdaptor> biomemap_adaptor;

	//Retrieve all flags
	auto isFlagged = []__host__(STPGeneratorOperation op, STPGeneratorOperation flag) -> bool {
		return (op & flag) != 0u;
	};
	const bool flag[3] = {
		isFlagged(operation, STPHeightfieldGenerator::HeightmapGeneration),
		isFlagged(operation, STPHeightfieldGenerator::Erosion),
		isFlagged(operation, STPHeightfieldGenerator::RenderingBufferGeneration)
	};

	STPcudaCheckErr(hipSetDevice(0));
	//setup phase
	//creating stream so cpu thread can calculate all chunks altogether
	optional<STPSmartStream> stream_buffer;
	hipStream_t stream;
	//we want the stream to not be blocked by default stream
	{
		unique_lock<mutex> stream_lock(this->StreamPool_lock);
		if (this->StreamPool.empty()) {
			//create a new stream
			stream_buffer.emplace(hipStreamNonBlocking);
		}
		else {
			//grab an exisiting stream
			stream_buffer.emplace(move(this->StreamPool.front()));
			this->StreamPool.pop();
		}
	}
	stream = *stream_buffer;

	//Flag: HeightmapGeneration
	if (flag[0]) {
		//generate a new heightmap using diversity generator and store it to the output later
		//copy biome map to device, and allocate heightmap
		STPFreeSlipFloatTextureBuffer::STPFreeSlipTextureData heightmap_data{ 1u, STPFreeSlipFloatTextureBuffer::STPFreeSlipTextureData::STPMemoryMode::WriteOnly, stream };
		STPFreeSlipSampleTextureBuffer::STPFreeSlipTextureData biomemap_data{ 1u, STPFreeSlipSampleTextureBuffer::STPFreeSlipTextureData::STPMemoryMode::ReadOnly, stream };
		try {
			heightmap_buffer.emplace(args.Heightmap32F, heightmap_data, this->TextureBufferAttr);
			biomemap_buffer.emplace(args.Biomemap, biomemap_data, this->TextureBufferAttr);
			biomemap_adaptor.emplace(this->FreeSlipTable(*biomemap_buffer));
		}
		catch (...) {
			exp = std::current_exception();
			goto freeUp;
		}
		this->generateHeightmap(*heightmap_buffer, *biomemap_adaptor, args.HeightmapOffset, stream);
	}
	else {
		//no generation, use existing
		STPFreeSlipFloatTextureBuffer::STPFreeSlipTextureData heightmap_data{ 1u, STPFreeSlipFloatTextureBuffer::STPFreeSlipTextureData::STPMemoryMode::ReadWrite, stream };
		try {
			heightmap_buffer.emplace(args.Heightmap32F, heightmap_data, this->TextureBufferAttr);
		}
		catch (...) {
			exp = std::current_exception();
			goto freeUp;
		}
		
	}

	if (flag[1] || flag[2]) {
		//prepare free-slip utility for heightmap
		heightmap_adaptor.emplace(this->FreeSlipTable(*heightmap_buffer));
		STPFreeSlipFloatManager heightmap_slip = (*heightmap_adaptor)(STPFreeSlipLocation::DeviceMemory);

		//Flag: Erosion
		if (flag[1]) {
			const unsigned erosionBrushCache_size = this->Heightfield_Setting_h.getErosionBrushSize() * (sizeof(int) + sizeof(float));
			STPcudaCheckErr(hipOccupancyMaxPotentialBlockSize(&Mingridsize, &blocksize, &performErosionKERNEL, erosionBrushCache_size));
			gridsize = (this->Heightfield_Setting_h.RainDropCount + blocksize - 1) / blocksize;

			//erode the heightmap, either from provided heightmap or generated previously
			performErosionKERNEL << <gridsize, blocksize, erosionBrushCache_size, stream >> > (heightmap_slip, this->Heightfield_Setting_d.get(), this->RNG_Map.get());
			STPcudaCheckErr(hipGetLastError());
		}

		//Flag: RenderingBufferGeneration
		if (flag[2]) {
			//allocate formation memory
			STPFreeSlipRenderTextureBuffer::STPFreeSlipTextureData heightfield_data{ 4u, STPFreeSlipRenderTextureBuffer::STPFreeSlipTextureData::STPMemoryMode::WriteOnly, stream };
			heightfield_buffer.emplace(args.Heightfield16UI, heightfield_data, this->TextureBufferAttr);

			auto det_cacheSize = []__host__ __device__(int blockSize) -> size_t {
				return (blockSize + 2u) * sizeof(float);
			};
			STPcudaCheckErr(hipOccupancyMaxPotentialBlockSizeVariableSMem(&Mingridsize, &blocksize, &generateRenderingBufferKERNEL, det_cacheSize));
			const uvec2 DimblockSize(32u, static_cast<unsigned int>(blocksize) / 32u),
				DimgridSize = (this->FreeSlipTable.getFreeSlipRange() + DimblockSize - 1u) / DimblockSize;

			//get free-slip util and memory
			unsigned short* heightfield_formatted_d = (*heightfield_buffer)(STPFreeSlipLocation::DeviceMemory);
			if (args.Heightfield16UI.size() > 1u) {
				//no need to do copy if freeslip is not enabled
				try {
					//this is the way to make sure normalmap is seamless, since the border is already in-sync with other chunks
					this->copyNeighbourEdgeOnly(heightfield_formatted_d, args.Heightfield16UI, this->TextureBufferAttr.TexturePixel, stream);
				}
				catch (...) {
					exp = std::current_exception();
					goto freeUp;
				}
				}
			//generate normalmap from heightmap and format into rendering buffer
			const uvec2 cacheBlockSize = DimblockSize + 2u;
			const unsigned int cacheSize = cacheBlockSize.x * cacheBlockSize.y * sizeof(float);
			generateRenderingBufferKERNEL << <dim3(DimgridSize.x, DimgridSize.y), dim3(DimblockSize.x, DimblockSize.y), cacheSize, stream >> > (heightmap_slip, this->Heightfield_Setting_h.Strength, heightfield_formatted_d);
			STPcudaCheckErr(hipGetLastError());
		}
	}
	
	//Store the result accordingly
	//copy the result back to the host
	freeUp:
	try {
		//it will call the destructor in texture buffer, and result will be copied back using CUDA stream
		//this operation is stream ordered
		heightmap_buffer.reset();
		heightfield_buffer.reset();
		biomemap_buffer.reset();
	}
	catch (...) {
		exp = std::current_exception();
	}

	//waiting for finish before release the stream back to the pool
	STPcudaCheckErr(hipStreamSynchronize(stream));
	{
		unique_lock<mutex> stream_lock(this->StreamPool_lock);
		this->StreamPool.emplace(move(*stream_buffer));
	}

	if (exp) {
		std::rethrow_exception(exp);
	}
}

__host__ void STPHeightfieldGenerator::setErosionIterationCUDA() {
	const unsigned int raindrop_count = this->Heightfield_Setting_h.RainDropCount;
	//determine launch parameters
	int Mingridsize, gridsize, blocksize;
	STPcudaCheckErr(hipOccupancyMaxPotentialBlockSize(&Mingridsize, &blocksize, &curandInitKERNEL));
	gridsize = (this->Heightfield_Setting_h.RainDropCount + blocksize - 1) / blocksize;

	//make sure all previous takes are finished
	STPcudaCheckErr(hipDeviceSynchronize());
	//when the raindrop count changes, we need to reallocate and regenerate the rng
	//the the number of rng = the number of the raindrop
	//such that each raindrop has independent rng
	//allocating spaces for rng storage array
	curandRNG* rng_cache;
	STPcudaCheckErr(hipMalloc(&rng_cache, sizeof(curandRNG) * raindrop_count));
	//and send to kernel
	curandInitKERNEL << <gridsize, blocksize >> > (rng_cache, this->Heightfield_Setting_h.Seed, raindrop_count);
	STPcudaCheckErr(hipGetLastError());
	STPcudaCheckErr(hipDeviceSynchronize());
	this->RNG_Map = unique_ptr_d<curandRNG>(rng_cache);
}

__host__ void STPHeightfieldGenerator::copyNeighbourEdgeOnly(unsigned short* device, const vector<unsigned short*>& source, size_t element_count, hipStream_t stream) const {
	typedef STPHeightfieldGenerator::STPEdgeArrangement STPEA;
	const uvec2& dimension = this->FreeSlipTable.getDimension();
	const unsigned int one_pixel_size = 4u * sizeof(unsigned short);
	const unsigned int pitch = dimension.x * one_pixel_size;
	const unsigned int horizontal_stripe_size = dimension.x * one_pixel_size;
	//we want to cut down the number of copy of column major matrix due to concern about cache
	/**
	* Out copy pattern:				It's more efficient than:
	 ---------------------			+-------------------+
	 |                   |			|                   |
	 |                   |			|                   |
	 |                   |			|                   |
	 |                   |			|                   |
	 ---------------------			+-------------------+
	*/
	//address offset of those situations, eliminate overlap of pixels
	const unsigned int right_vertical_wholerow = (dimension.x - 1u) * 4u,
		left_vertical_skipfirstrow = right_vertical_wholerow + 4u,
		right_vertical_skipfirstrow = left_vertical_skipfirstrow + right_vertical_wholerow,
		bottom_horizontal = dimension.x * 4u * (dimension.y - 1u);

	for (int i = 0; i < source.size(); i++) {
		auto perform_copy = [device, stream, map = source[i], &pitch]__host__(size_t start, size_t width_byte, size_t height) -> void {
			STPcudaCheckErr(hipMemcpy2DAsync(device + start, pitch, map + start, pitch, width_byte, height, hipMemcpyHostToDevice, stream));
		};

		switch (this->EdgeArrangementTable[i]) {
		case STPEA::TOP_LEFT_CORNER:
			//------------
			//|
			//|
			//|
			//|
			perform_copy(0u, horizontal_stripe_size, 1u);
			perform_copy(left_vertical_skipfirstrow, one_pixel_size, dimension.y - 1u);
			break;
		case STPEA::TOP_RIGHT_CORNER:
			//-------------
			//            |
			//            |
			//            |
			//            |
			perform_copy(0u, horizontal_stripe_size, 1u);
			perform_copy(right_vertical_skipfirstrow, one_pixel_size, dimension.y - 1u);
			break;
		case STPEA::BOTTOM_LEFT_CORNER:
			//|
			//|
			//|
			//|
			//-------------
			perform_copy(bottom_horizontal, horizontal_stripe_size, 1u);
			perform_copy(0u, one_pixel_size, dimension.y - 1u);
			break;
		case STPEA::BOTTOM_RIGHT_CORNER:
			//             |
			//             |
			//             |
			//             |
			//--------------
			perform_copy(bottom_horizontal, horizontal_stripe_size, 1u);
			perform_copy(right_vertical_wholerow, one_pixel_size, dimension.y - 1u);
			break;
		case STPEA::TOP_HORIZONTAL_STRIP:
			//--------------
			//
			//
			//
			//
			perform_copy(0u, horizontal_stripe_size, 1u);
			break;
		case STPEA::BOTTOM_HORIZONTAL_STRIP:
			//
			//
			//
			//
			//--------------
			perform_copy(bottom_horizontal, horizontal_stripe_size, 1u);
			break;
		case STPEA::LEFT_VERTICAL_STRIP:
			//|
			//|
			//|
			//|
			//|
			perform_copy(0u, one_pixel_size, dimension.y);
			break;
		case STPEA::RIGHT_VERTICAL_STRIP:
			//             |
			//             |
			//             |
			//             |
			//             |
			perform_copy(right_vertical_wholerow, one_pixel_size, dimension.y);
			break;
		default:
			//skip every non-edge chunk
			break;
		}
		device += element_count * 4u;
	}
}

__host__ void STPHeightfieldGenerator::initEdgeArrangementTable() {
	typedef STPHeightfieldGenerator::STPEdgeArrangement STPEA;
	const uvec2& freeslip_chunk = this->FreeSlipTable.getFreeSlipChunk();
	const unsigned int num_chunk = freeslip_chunk.x * freeslip_chunk.y;
	if (num_chunk == 1u) {
		//if freeslip logic is not turned on, there's no need to do copy
		//since edge is calculated by neighbour chunks, but without freeslip there's no "other" chunks
		//so the chunk itself needs to compute the border during rendering buffer generation
		return;
	}

	//allocate space
	this->EdgeArrangementTable = make_unique<STPEA[]>(num_chunk);
	for (unsigned int chunkID = 0u; chunkID < num_chunk; chunkID++) {
		STPEA& current_entry = this->EdgeArrangementTable[chunkID];
		const uvec2 chunkCoord(chunkID % freeslip_chunk.x, chunkID / freeslip_chunk.x);

		//some basic boolean logic to determine our "frame"
		if (chunkCoord.x == 0u) {
			if (chunkCoord.y == 0u) {
				current_entry = STPEA::TOP_LEFT_CORNER;
				continue;
			}
			if (chunkCoord.y == freeslip_chunk.y - 1u) {
				current_entry = STPEA::BOTTOM_LEFT_CORNER;
				continue;
			}
			current_entry = STPEA::LEFT_VERTICAL_STRIP;
			continue;
		}
		if (chunkCoord.x == freeslip_chunk.x - 1u) {
			if (chunkCoord.y == 0u) {
				current_entry = STPEA::TOP_RIGHT_CORNER;
				continue;
			}
			if (chunkCoord.y == freeslip_chunk.y - 1u) {
				current_entry = STPEA::BOTTOM_RIGHT_CORNER;
				continue;
			}
			current_entry = STPEA::RIGHT_VERTICAL_STRIP;
			continue;
		}

		if (chunkCoord.y == 0u) {
			current_entry = STPEA::TOP_HORIZONTAL_STRIP;
			continue;
		}
		if (chunkCoord.y == freeslip_chunk.y - 1u) {
			current_entry = STPEA::BOTTOM_HORIZONTAL_STRIP;
			continue;
		}

		//we can safely ignore edge that's not an edge chunk
		current_entry = STPEA::NOT_AN_EDGE;
	}
}

__global__ void curandInitKERNEL(STPHeightfieldGenerator::curandRNG* rng, unsigned long long seed, unsigned int raindrop_count) {
	//current working index
	const unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index >= raindrop_count) {
		return;
	}

	//the same seed but we are looking for different sequence
	hiprand_init(seed, static_cast<unsigned long long>(index), 0, &rng[index]);
}

__global__ void performErosionKERNEL(STPFreeSlipFloatManager heightmap_storage, const SuperTerrainPlus::STPEnvironment::STPHeightfieldSetting* heightfield_settings, STPHeightfieldGenerator::curandRNG* rng) {
	//current working index
	const unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index >= heightfield_settings->RainDropCount) {
		return;
	}

	//convert to (base, dimension - 1]
	//range: dimension
	//Generate the raindrop at the central chunk only
	__shared__ uvec2 base;
	__shared__ uvec2 range;
	if (threadIdx.x == 0u) {
		const uvec2& dimension = heightmap_storage.Data->Dimension;

		base = dimension - 1u,
		range = (heightmap_storage.Data->FreeSlipChunk / 2u) * dimension;
	}
	__syncthreads();

	//generating random location
	//first we generate the number (0.0f, 1.0f]
	vec2 initPos = vec2(hiprand_uniform(&rng[index]), hiprand_uniform(&rng[index]));
	//range convertion
	initPos *= base;
	initPos += range;

	//spawn in the raindrop
	STPRainDrop droplet(initPos, heightfield_settings->initWaterVolume, heightfield_settings->initSpeed);
	droplet.Erode(static_cast<const SuperTerrainPlus::STPEnvironment::STPRainDropSetting*>(heightfield_settings), heightmap_storage);
}

__global__ void generateRenderingBufferKERNEL(STPFreeSlipFloatManager heightmap, float strength, unsigned short* heightfield) {
	//the current working pixel
	const uvec2 block = uvec2(blockIdx.x * blockDim.x, blockIdx.y * blockDim.y),
		local_thread = uvec2(threadIdx.x, threadIdx.y),
		thread = local_thread + block;
	const unsigned int threadperblock = blockDim.x * blockDim.y;
	const uvec2& freeslip_range = heightmap.Data->FreeSlipRange;
	if (thread.x >= freeslip_range.x || thread.y >= freeslip_range.y) {
		return;
	}

	const uvec2& dimension = heightmap.Data->FreeSlipRange;
	auto clamp = []__device__(int val, int lower, int upper) -> int {
		return max(lower, min(val, upper));
	};
	auto float2short = []__device__(float input) -> unsigned short {
		return static_cast<unsigned short>(input * 65535u);
	};
	 
	//Cache heightmap the current thread block needs since each pixel is accessed upto 9 times.
	extern __shared__ float heightmapCache[];
	//each thread needs to access a 3x3 matrix around the current pixel, so we need to take the edge into consideration
	const uvec2 cacheSize = uvec2(blockDim.x, blockDim.y) + 2u;
	unsigned int iteration = 0u;
	const unsigned int cacheSize_total = cacheSize.x * cacheSize.y;

	while (iteration < cacheSize_total) {
		const unsigned int cacheIdx = (threadIdx.x + blockDim.x * threadIdx.y) + iteration;
		const uvec2 worker = block + uvec2(cacheIdx % cacheSize.x, cacheIdx / cacheSize.x);
		const unsigned int workerIdx = clamp((worker.x - 1u), 0, dimension.x - 1u) + clamp((worker.y - 1u), 0, dimension.y - 1u) * dimension.x;

		if (cacheIdx < cacheSize_total) {
			//make sure index don't get out of bound
			//start caching from (x-1, y-1) until (x+1, y+1)
			heightmapCache[cacheIdx] = heightmap[workerIdx];
		}
		//warp around to reuse some threads to finish all compute
		iteration += threadperblock;
	}
	__syncthreads();

	if ((heightmap.Data->FreeSlipChunk.x * heightmap.Data->FreeSlipChunk.y) > 1 && 
		(thread.x == 0 || thread.y == 0 || thread.x == freeslip_range.x - 1 || thread.y == freeslip_range.y - 1)) {
		//if freeslip is not turned on, we need to calculate the edge pixel for this chunk
		//otherwise, do not touch the border pixel since border pixel is calculated seamlessly by other chunks
		return;
	}
	//load the cells from heightmap, remember the height map only contains one color channel
	//using Sobel fitering
	//Cache index
	const uvec2 cache = local_thread + 1u;
	float cell[8];
	cell[0] = heightmapCache[(cache.x - 1) + (cache.y - 1) * cacheSize.x];
	cell[1] = heightmapCache[cache.x + (cache.y - 1) * cacheSize.x];
	cell[2] = heightmapCache[(cache.x + 1) + (cache.y - 1) * cacheSize.x];
	cell[3] = heightmapCache[(cache.x - 1) + cache.y * cacheSize.x];
	cell[4] = heightmapCache[(cache.x + 1) + cache.y * cacheSize.x];
	cell[5] = heightmapCache[(cache.x - 1) + (cache.y + 1) * cacheSize.x];
	cell[6] = heightmapCache[cache.x + (cache.y + 1) * cacheSize.x];
	cell[7] = heightmapCache[(cache.x + 1) + (cache.y + 1) * cacheSize.x];
	//apply the filtering kernel matrix
	vec3 normal;
	normal.z = 1.0f / strength;
	normal.x = cell[0] + 2 * cell[3] + cell[5] - (cell[2] + 2 * cell[4] + cell[7]);
	normal.y = cell[0] + 2 * cell[1] + cell[2] - (cell[5] + 2 * cell[6] + cell[7]);
	//normalize
	normal = glm::normalize(normal);
	//clamp to [0,1], was [-1,1]
	normal = (glm::clamp(normal, -1.0f, 1.0f) + 1.0f) / 2.0f;
	
	//copy to the output, RGBA32F
	const unsigned int index = heightmap(thread.x + thread.y * dimension.x);
	heightfield[index * 4] = float2short(normal.x);//R
	heightfield[index * 4 + 1] = float2short(normal.y);//G
	heightfield[index * 4 + 2] = float2short(normal.z);//B
	heightfield[index * 4 + 3] = float2short(heightmapCache[cache.x + cache.y * cacheSize.x]);//A
}