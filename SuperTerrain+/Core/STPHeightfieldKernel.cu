#include "hip/hip_runtime.h"
#include <SuperTerrain+/GPGPU/STPHeightfieldKernel.cuh>

#include <SuperTerrain+/Utility/STPSmartDeviceMemory.tpp>
//Error
#include <SuperTerrain+/Utility/STPDeviceErrorHandler.h>

/* --------- Kernel Declaration ----------- */

using namespace SuperTerrainPlus;
using namespace SuperTerrainPlus::STPCompute;

#include <glm/geometric.hpp>

using glm::uvec2;

__global__ static void curandInitKERNEL(STPHeightfieldKernel::STPcurand_t*, unsigned long long, unsigned int);

__global__ static void initGlobalLocalIndexKERNEL(unsigned int*, uvec2, uvec2, uvec2);

__global__ static void hydraulicErosionKERNEL(STPFreeSlipFloatManager, const STPEnvironment::STPHeightfieldSetting*, STPHeightfieldKernel::STPcurand_t*);

__global__ static void texture32Fto16KERNEL(float*, unsigned short*, uvec2);

__host__ STPHeightfieldKernel::STPcurand_arr STPHeightfieldKernel::curandInit(unsigned long long seed, unsigned int count) {
	//determine launch parameters
	int Mingridsize, gridsize, blocksize;
	STPcudaCheckErr(hipOccupancyMaxPotentialBlockSize(&Mingridsize, &blocksize, &curandInitKERNEL));
	gridsize = (count + blocksize - 1) / blocksize;

	//allocating spaces for rng storage array
	STPcurand_arr rng = STPSmartDeviceMemory::makeDevice<STPcurand_t[]>(count);
	//and send to kernel to init rng sequences
	curandInitKERNEL << <gridsize, blocksize >> > (rng.get(), seed, count);
	STPcudaCheckErr(hipGetLastError());

	return rng;
}

__host__ STPHeightfieldKernel::STPIndexTable STPHeightfieldKernel::initGlobalLocalIndex(uvec2 chunkRange, uvec2 tableSize, uvec2 mapSize) {
	const size_t index_count = tableSize.x * tableSize.y;
	//launch parameters
	int Mingridsize, blocksize;
	STPcudaCheckErr(hipOccupancyMaxPotentialBlockSize(&Mingridsize, &blocksize, &initGlobalLocalIndexKERNEL));
	const uvec2 Dimblocksize(32u, static_cast<unsigned int>(blocksize) / 32u),
		Dimgridsize = (tableSize + Dimblocksize - 1u) / Dimblocksize;

	//allocation
	STPIndexTable indexTable = STPSmartDeviceMemory::makeDevice<unsigned int[]>(index_count);
	//compute
	initGlobalLocalIndexKERNEL << <dim3(Dimgridsize.x, Dimgridsize.y), dim3(Dimblocksize.x, Dimblocksize.y) >> > (
		indexTable.get(), chunkRange, tableSize, mapSize);
	STPcudaCheckErr(hipGetLastError());

	return indexTable;
}

__host__ void STPHeightfieldKernel::hydraulicErosion
	(STPFreeSlipFloatManager heightmap_storage, const STPEnvironment::STPHeightfieldSetting* heightfield_settings, 
		unsigned int brush_size, unsigned int raindrop_count, STPcurand_t* rng, hipStream_t stream) {
	//brush contains two components: weights (float) and indices (int)
	const unsigned int erosionBrushCache_size = brush_size * (sizeof(int) + sizeof(float));
	//launc para
	int Mingridsize, gridsize, blocksize;
	STPcudaCheckErr(hipOccupancyMaxPotentialBlockSize(&Mingridsize, &blocksize, &hydraulicErosionKERNEL, erosionBrushCache_size));
	gridsize = (raindrop_count + blocksize - 1) / blocksize;

	//erode the heightmap
	hydraulicErosionKERNEL << <gridsize, blocksize, erosionBrushCache_size, stream >> > (heightmap_storage, heightfield_settings, rng);
	STPcudaCheckErr(hipGetLastError());
}

__host__ void STPHeightfieldKernel::texture32Fto16(float* input, unsigned short* output, uvec2 dimension, unsigned int channel, hipStream_t stream) {
	const uvec2 totalDimension = dimension * channel;

	int Mingridsize, blocksize;
	STPcudaCheckErr(hipOccupancyMaxPotentialBlockSize(&Mingridsize, &blocksize, &texture32Fto16KERNEL));
	const uvec2 Dimblocksize(32u, static_cast<unsigned int>(blocksize) / 32u),
		Dimgridsize = (totalDimension + Dimblocksize - 1u) / Dimblocksize;

	//compute
	texture32Fto16KERNEL << <dim3(Dimgridsize.x, Dimgridsize.y), dim3(Dimblocksize.x, Dimblocksize.y), 0, stream >> > (input, output, totalDimension);
	STPcudaCheckErr(hipGetLastError());
}

/* --------- Kernel Definition ----------- */

#include <>

using glm::vec2;

__global__ void curandInitKERNEL(STPHeightfieldKernel::STPcurand_t* rng, unsigned long long seed, unsigned int count) {
	//current working index
	const unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index >= count) {
		return;
	}

	//the same seed but we are looking for different sequence
	hiprand_init(seed, static_cast<unsigned long long>(index), 0, &rng[index]);
}

__global__ void initGlobalLocalIndexKERNEL(unsigned int* output, uvec2 chunkRange, uvec2 tableSize, uvec2 mapSize) {
	//current pixel
	const unsigned int x = (blockIdx.x * blockDim.x) + threadIdx.x,
		y = (blockIdx.y * blockDim.y) + threadIdx.y,
		rowCount = tableSize.x,
		globalidx = x + y * rowCount;
	if (x >= tableSize.x || y >= tableSize.y) {
		return;
	}

	//simple maths
	const uvec2 globalPos = uvec2(globalidx - y * rowCount, y);
	const uvec2 chunkPos = globalPos / mapSize;//non-negative integer division is a floor
	const uvec2 localPos = globalPos - chunkPos * mapSize;

	output[globalidx] = (chunkPos.x + chunkRange.x * chunkPos.y) * mapSize.x * mapSize.y + (localPos.x + mapSize.x * localPos.y);
}

//It's raining
#include <SuperTerrain+/GPGPU/STPRainDrop.cuh>

__global__ void hydraulicErosionKERNEL
	(STPFreeSlipFloatManager heightmap_storage, const STPEnvironment::STPHeightfieldSetting* heightfield_settings, STPHeightfieldKernel::STPcurand_t* rng) {
	//current working index
	const unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index >= heightfield_settings->RainDropCount) {
		return;
	}

	//convert to (base, dimension - 1]
	//range: dimension
	//Generate the raindrop at the central chunk only
	__shared__ uvec2 base;
	__shared__ uvec2 range;
	if (threadIdx.x == 0u) {
		const uvec2& dimension = heightmap_storage.Data->Dimension;

		base = dimension - 1u,
			range = (heightmap_storage.Data->FreeSlipChunk / 2u) * dimension;
	}
	__syncthreads();

	//generating random location
	//first we generate the number (0.0f, 1.0f]
	vec2 initPos = vec2(hiprand_uniform(&rng[index]), hiprand_uniform(&rng[index]));
	//range convertion
	initPos *= base;
	initPos += range;

	//spawn the raindrop
	STPRainDrop droplet(initPos, heightfield_settings->initWaterVolume, heightfield_settings->initSpeed);
	droplet.Erode(static_cast<const STPEnvironment::STPRainDropSetting*>(heightfield_settings), heightmap_storage);
}

#include <limits>
constexpr static unsigned short FP32toUINT16constant = std::numeric_limits<unsigned short>::max();

__global__ void texture32Fto16KERNEL(float* input, unsigned short* output, uvec2 dimension) {
	//the current working pixel
	const unsigned int x = (blockIdx.x * blockDim.x) + threadIdx.x,
		y = (blockIdx.y * blockDim.y) + threadIdx.y,
		index = x + y * dimension.x;
	//range check
	if (x >= dimension.x || y >= dimension.y) {
		return;
	}

	output[index] = static_cast<unsigned short>(glm::clamp(input[index], 0.0f, 1.0f) * FP32toUINT16constant);
}