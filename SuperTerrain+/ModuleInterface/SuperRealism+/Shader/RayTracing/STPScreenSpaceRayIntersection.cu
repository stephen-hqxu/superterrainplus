#include "hip/hip_runtime.h"
#include "STPScreenSpaceRayIntersection.cuh"

#include "STPFragmentUtility.cuh"
#include "STPVectorUtility.cuh"
#include "STPInstanceIDCoder.cuh"

//OptiX
#include <optix.h>

//System
#include <cuda/std/limits>

using namespace SuperTerrainPlus::STPRealism;

extern __constant__ STPScreenSpaceRayIntersectionData SSRIData;

//Ray data passed between each shader
struct STPSSRIPayload {
public:

	//The total number of payload register used
	constexpr static unsigned int DataCount = 6u;

	unsigned int PrimitiveID;
	float3 Position;
	float2 UV;

};

__device__ __forceinline__ static STPSSRIPayload traceIntersection(const float3& origin, const float3& direction, unsigned int mask) {
	//initialise all payloads as undefined
	unsigned int p[STPSSRIPayload::DataCount];
	for (unsigned int i = 0u; i < STPSSRIPayload::DataCount; i++) {
		p[i] = optixUndefinedValue();
	}
	//unlike rasterisation, back face culling may incur performance penalty
	optixTrace(SSRIData.Handle,
		origin, direction,
		0.0f, 1e16f, 0.0f,
		mask, OPTIX_RAY_FLAG_DISABLE_ANYHIT,
		0u, 0u, 0u,
		p[0], p[1], p[2], p[3], p[4], p[5]
	);

	//read payloads
	STPSSRIPayload data;
	data.PrimitiveID = p[0];
	data.Position = make_float3(
		__uint_as_float(p[1]),
		__uint_as_float(p[2]),
		__uint_as_float(p[3])
	);
	data.UV = make_float2(
		__uint_as_float(p[4]),
		__uint_as_float(p[5])
	);
	return data;
}

__device__ __forceinline__ static void setPrimitiveData(const STPSSRIPayload& data) {
	//record intersection information
	optixSetPayload_0(data.PrimitiveID);
	optixSetPayload_1(__float_as_uint(data.Position.x));
	optixSetPayload_2(__float_as_uint(data.Position.y));
	optixSetPayload_3(__float_as_uint(data.Position.z));
	optixSetPayload_4(__float_as_uint(data.UV.x));
	optixSetPayload_5(__float_as_uint(data.UV.y));
}

__device__ __forceinline__ static void setEnvironmentData() {
	//primitiveID zero is reserved for no intersection,
	//which either denotes an environment pixel, or ray is invisible
	optixSetPayload_0(STPScreenSpaceRayIntersectionData::EnvironmentRayID);
	optixSetPayload_1(optixUndefinedValue());
	optixSetPayload_2(optixUndefinedValue());
	optixSetPayload_3(optixUndefinedValue());
	optixSetPayload_4(optixUndefinedValue());
	optixSetPayload_5(optixUndefinedValue());
}

__global__ void __raygen__launchScreenSpaceRay() {
	//we are always using 2D launch, so ignore the third component
	const uint2 idx = make_uint2(optixGetLaunchIndex()),
		dim = make_uint2(optixGetLaunchDimensions());

	const float2 texCoord = make_float2(idx);
	//recover fragment values from texture, using un-normalised UV
	const auto stencil = surf2Dread<unsigned char>(SSRIData.SSStencil, texCoord.x * sizeof(unsigned char), texCoord.y);
	const auto ray_depth = tex2D<float>(SSRIData.SSRayDepth, texCoord.x, texCoord.y);
	const auto ray_dir =
		make_float3(tex2D<float4>(SSRIData.SSRayDirection, texCoord.x, texCoord.y)) * 2.0f - 1.0f;
	//calculate ray origin using normalised UV
	const float2 uv = STPFragmentUtility::calcTextureCoordinate(idx, dim);
	const float3 ray_ori = STPFragmentUtility::reconstructDepthToWorld(SSRIData.InvProjectionView, ray_depth, uv);

	//stencil test
	const unsigned char stencil_result = STPScreenSpaceRayIntersectionData::RayVisibilityMask & stencil;
	const unsigned int rayVisibility = stencil_result == 0u ? 0x00u : 0xFFu;

	//start the magic
	//doing a branch based on stencil test on trace function gives poor performance,
	//instead we can interpret stencil as ray visibility.
	const STPSSRIPayload data = traceIntersection(ray_ori, ray_dir, rayVisibility);
	
	//store to texture
	if (!rayVisibility) {
		//record result to texture only when ray is visible
		//mainly to save memory bandwidth, but for stencil buffer we preserve the original stencil if stencil test fails
		return;
	}
	//store stencil result
	surf2Dwrite(static_cast<unsigned char>(stencil_result | data.PrimitiveID), SSRIData.SSStencil, texCoord.x * sizeof(unsigned char), texCoord.y);

	if (data.PrimitiveID == STPScreenSpaceRayIntersectionData::EnvironmentRayID) {
		//environment ray has no vertex data
		return;
	}
	const uint2 pixel_uv = make_uint2(data.UV * cuda::std::numeric_limits<unsigned short>::max());
	surf2Dwrite(make_float4(data.Position, 1.0f), SSRIData.GPosition, texCoord.x * sizeof(float4), texCoord.y);
	surf2Dwrite(make_ushort2(pixel_uv.x, pixel_uv.y), SSRIData.GTextureCoordinate, texCoord.x * sizeof(ushort2), texCoord.y);
}

__global__ void __closesthit__recordPrimitiveIntersection() {
	const auto* const data = reinterpret_cast<const STPScreenSpaceRayIntersectionData::STPPrimitiveHitData*>(optixGetSbtDataPointer()); 
	//read primitive vertex data
	const auto [objectID, instanceID] = STPInstanceIDCoder::decode(optixGetInstanceId());
	const uint3& attributeIdx = data->PrimitiveIndex[objectID][instanceID][optixGetPrimitiveIndex()];
	//grab data of each vertex
	float3 position[3];
	float2 uv[3];
	const float* const baseVertex = data->PrimitiveVertex[objectID][instanceID];
	for (unsigned int i = 0u; i < 3u; i++) {
		const float* const vertex =
			baseVertex + getByIndex(attributeIdx, i) * STPScreenSpaceRayIntersectionData::STPPrimitiveHitData::AttributeStride;
		position[i] = make_float3(vertex[0], vertex[1], vertex[2]);
		uv[i] = make_float2(vertex[3], vertex[4]);
	}

	//vertex interpolation
	STPSSRIPayload prd;
	const float2 bary = optixGetTriangleBarycentrics();
	prd.PrimitiveID = objectID;
	//remember to perform space conversion for position
	prd.Position = optixTransformPointFromObjectToWorldSpace(
		STPFragmentUtility::barycentricInterpolation(bary, position[0], position[1], position[2]));
	prd.UV = STPFragmentUtility::barycentricInterpolation(bary, uv[0], uv[1], uv[2]);
	//store data to payload
	setPrimitiveData(prd);
}

__global__ void __miss__recordEnvironmentIntersection() {
	setEnvironmentData();
}