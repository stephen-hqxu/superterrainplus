#include "hip/hip_runtime.h"
//-------------------- Constants ----------------------------
constexpr static float 
	//Control the gradient change, greater decreases gradient in general.
	GradientBias = 5.5f, 
	//Control the distance of pixels in the filter kernel.
	KernelRadius = 2.5f, 
	//Control the scale of noise, greater denominator gives smoother noise.
	NoiseScale = 91.5f,
	//Control how much the noise can affect the height value
	NoiseContribution = 0.15f,
	//A.k.a. altitude, but it does not have to be the same as altitude of the terrain.
	//Control sensitivity of gradient responds to altitude change.
	HeightFactor = 875.5f;

//Simplex noise fractal settings
constexpr static float
	Per = 0.7f,
	Lac = 1.88f;

//--------------------- Program Start ------------------------
#include "./Script/STPCommonGenerator.cuh"

//SuperAlgorithm+ Device library
#include <SuperAlgorithm+Device/STPTextureSplatRuleWrapper.cuh>

#include <SuperTerrain+/Utility/STPDeviceLaunchSetup.cuh>

using namespace SuperTerrainPlus::STPAlgorithm;
using SuperTerrainPlus::STPSample_t, SuperTerrainPlus::STPRegion_t;

namespace STPTI = SuperTerrainPlus::STPDiversity::STPTextureInformation;

__constant__ STPTI::STPSplatRuleDatabase SplatDatabase[1];

//A simple 2x2 kernel
constexpr static unsigned int GradientSize = 4u;
constexpr static int2 GradientKernel[GradientSize] = {
	int2{ 0, -1 },//top, 0
	int2{ -1, 0 },//left, 1
	int2{ 1, 0 },//right, 2
	int2{ 0, 1 }//bottom, 3
};

/**
 * @brief Generate some simplex noise for an input.
 * @param x The x coordinate
 * @param y The y coordinate
 * @param offset The noise offset
 * @return The normalised noise with noise contribution applied.
*/
__device__ static float generateNoise(unsigned int, unsigned int, float2);

//--------------------- Definition --------------------------

using namespace STPCommonGenerator;

/**
 * @brief Launch kernel to start splatmap generation.
 * All texture objects are non-layered 2D.
 * @param biomemap_tex The biomemap texture input
 * @param heightmap_tex The heightmap texture input
 * @param splatmap_surf The splatmap surface output
 * @param splat_info Information about the generation.
*/
__global__ void generateTextureSplatmap(const hipTextureObject_t biomemap_tex, const hipTextureObject_t heightmap_tex,
	const hipSurfaceObject_t splatmap_surf, const STPTI::STPSplatGeneratorInformation splat_info) {
	//block is in 2D, so threadIdx.z is always 0 and blockDim.z is always 1
	const auto [x, y, z] = SuperTerrainPlus::STPDeviceLaunchSetup::calcThreadIndex<3u>();
	if (x >= Dimension->x || y >= Dimension->y || z >= splat_info.LocalCount) {
		return;
	}
	//working pixel
	//we need to convert z-coord of thread to chunk local ID
	const STPTI::STPSplatGeneratorInformation::STPLocalChunkInformation& local_info = splat_info.RequestingLocalInfo[z];

	//coordinates are un-normalised
	const uint2 SamplingPosition = make_uint2(
		x + Dimension->x * local_info.LocalChunkCoordinateX,
		y + Dimension->y * local_info.LocalChunkCoordinateY
	);

	float cell[GradientSize];
	//calculate heightmap gradient
	for (unsigned int i = 0u; i < GradientSize; i++) {
		const int2& currentKernel = GradientKernel[i];
		const float2 offsetUV = make_float2(
			currentKernel.x * KernelRadius,
			currentKernel.y * KernelRadius
		);
		const float2 SamplingUV = make_float2(
			SamplingPosition.x + offsetUV.x,
			SamplingPosition.y + offsetUV.y
		);
		//sample this heightmap value
		cell[i] = tex2D<float>(heightmap_tex, SamplingUV.x, SamplingUV.y) * HeightFactor;
	}

	//calculate gradient using a very simple 2x2 filter, ranged [-1,1]
	const float gradient[3] = {
		cell[0] - cell[3],
		GradientBias,
		cell[1] - cell[2]
	};
	const float slopFactor = 1.0f - (gradient[1] * rnormf(3, gradient));

	//add some simplex noise to the slopFactor and height value, reminder: range is [-1,1]
	const float noise = generateNoise(x, y, make_float2(local_info.ChunkMapOffsetX, local_info.ChunkMapOffsetY));
	//get information about the current position
	const STPSample_t biome = tex2D<STPSample_t>(biomemap_tex, SamplingPosition.x, SamplingPosition.y);
	const float height = __saturatef(tex2D<float>(heightmap_tex, SamplingPosition.x, SamplingPosition.y) + noise);

	const STPTextureSplatRuleWrapper splatWrapper(*SplatDatabase);
	const STPTI::STPSplatRegistry* const registry = splatWrapper.findSplatRegistry(biome);
	//get regions, we define gradient region outweighs altitude region if they overlap
	unsigned int region = splatWrapper.gradientRegion(registry, slopFactor, height);
	if (region == STPTextureSplatRuleWrapper::NoRegion) {
		//no gradient region is being defined, switch to altitude region
		region = splatWrapper.altitudeRegion(registry, height);
		//we don't need to check for null altitude region, if there is none, there is none...
	}
	//write whatever region to the splatmap
	//out-of-boundary write will be caught by CUDA (safely) and will crash the program with error
	surf2Dwrite(static_cast<STPRegion_t>(region), splatmap_surf, SamplingPosition.x * sizeof(STPRegion_t),
		SamplingPosition.y, hipBoundaryModeTrap);
}

__device__ float generateNoise(const unsigned int x, const unsigned int y, const float2 offset) {
	//use simplex noise to generate fractals
	STPSimplexNoise::STPFractalSimplexInformation fractal_info = { };
	fractal_info.Persistence = Per;
	fractal_info.Lacunarity = Lac;
	fractal_info.Octave = 3u;
	fractal_info.Scale = NoiseScale;
	fractal_info.Offset = offset;
	fractal_info.HalfDimension = *HalfDimension;

	return STPSimplexNoise::simplex2DFractal(*Permutation, 1.0f * x, 1.0f * y, fractal_info) * NoiseContribution;
}