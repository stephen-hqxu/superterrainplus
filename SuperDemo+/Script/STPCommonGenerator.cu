#include "hip/hip_runtime.h"
#include "./Script/STPCommonGenerator.cuh"

using namespace SuperTerrainPlus::STPCompute;

//Those variables are defined in this source file, only
__constant__ uint2 STPCommonGenerator::Dimension[1];
__constant__ float2 STPCommonGenerator::HalfDimension[1];
__constant__ uint2 STPCommonGenerator::RenderedDimension[1];

__constant__ STPPermutation STPCommonGenerator::Permutation[1];